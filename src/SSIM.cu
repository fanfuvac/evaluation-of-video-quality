#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include "main.h"  
#include "ssim.cuh"  
//#include "..\..\Video_comparsion\Video_comparsion\PSNR.h"   
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <errno.h>
#include <stdio.h>
using namespace std;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__host__ __device__ double countAvg(unsigned char * data);
__host__ __device__ double countVariance(unsigned char * data, double avg);
__host__ __device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2);
__host__ __device__ void getRect(unsigned char* data, int start, int width, unsigned char * out);
__host__ __device__ double countRectangle(unsigned char * data1, unsigned char * data2);
double countRes(double * tmpRes, int count);

double countSSIM(unsigned char * datain1, unsigned char * datain2, unsigned char * dataC1, unsigned char * dataC2, unsigned char * rects1, unsigned char * rects2, int size, int width, double*& results) {
	//unsigned char * data1 = (unsigned char*)datain1;
	//unsigned char * data2 = (unsigned char*)datain2;
	hipError_t cudaStatus;
	//double * tmpRes = new double[size];


	/*
	if (data1==0 or data2==0){
	//return NULL;
	//cout<<"error in allocation"<<endl;
	}*/
	/*getLuma(datain1, data1, size);
	getLuma(datain2, data2, size);*/
	/*unsigned char * rect1 = new unsigned char[RECT_SIZE];
	unsigned char * rect2 = new unsigned char[RECT_SIZE];
	int k = 0;*/



	//#pragma omp parallel
	//nthreads = omp_get_num_threads();


	cudaStatus = hipMemcpy(dataC1, datain1, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return NULL;
	}
	cudaStatus = hipMemcpy(dataC2, datain2, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return NULL;
	}


	//	#pragma omp parallel for schedule(static, 20)
	int rectCount = (size / width - RECT_SQRT)*(width - RECT_SQRT) / SKIP_SIZE / SKIP_SIZE / THREADS*THREADS;
	int blocks = rectCount / THREADS;
	if (rectCount< (size / width - RECT_SQRT)*(width - RECT_SQRT) / SKIP_SIZE / SKIP_SIZE) {
		blocks = rectCount / THREADS + 1;
		rectCount = (size / width - RECT_SQRT)*(width - RECT_SQRT) / SKIP_SIZE / SKIP_SIZE;
	}
	countRectangleKernel << <blocks, THREADS >> >(dataC1, dataC2, rects1, rects2, results, size, width); //FIXME - need to adjust size to count up to THREADS last rectangles!!
																										 //hipDeviceSynchronize();
																										 /*for (int i = 0; i < size / width - RECT_SQRT; i += SKIP_SIZE) {

																										 for (int j = 0; j < width - RECT_SQRT; j += SKIP_SIZE, k++) {
																										 //for (int i = 0; i < size-(RECT_SQRT-1)*width; i+=SKIP_SIZE) {

																										 //if (tmpRes[k] < 0) cout << "low result: " << i<< ": " << j<< " :" << tmpRes[k] << endl;

																										 }
																										 }


																										 double res = countRes(tmpRes, k);
																										 */



	double * resultsOut = new double[rectCount];
	cudaStatus = hipMemcpy(resultsOut, results, rectCount * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return NULL;
	}
	double output = countRes(resultsOut, rectCount);

	delete resultsOut;
	//cout << output << endl;
	return output;
}





//return one rectangle with RECT_SIZE pixels
__host__ __device__ void getRect(unsigned char* data, int start, int width, unsigned char * out) {
#if defined(__CUDA_ARCH__)
	for (int i = 0; i<RECT_SQRT; i++) {
		for (int j = 0; j < RECT_SQRT; j++) {
			out[i*RECT_SQRT + j] = data[start + i*width + j];
		}
		//hipMemcpy(out + i*RECT_SQRT, data + start + i*width, RECT_SQRT, hipMemcpyDeviceToDevice);
	}
#else
	for (int i = 0; i<RECT_SQRT; i++) {
		memcpy(out + i*RECT_SQRT, data + start + i*width, RECT_SQRT);
	}
#endif
	//return out;
}

//count ssim of one rectangle with RECT_SIZE pixels
__host__ __device__ double countRectangle(unsigned char * data1, unsigned char * data2) {

	double avg1 = countAvg(data1);
	double avg2 = countAvg(data2);

	double var1 = countVariance(data1, avg1);
	double var2 = countVariance(data2, avg2);

	double cov = countCovariance(data1, data2, avg1, avg2);


	double ssim = ((2 * avg1*avg2 + C1)*(2 * cov + C2)) / ((avg1*avg1 + avg2*avg2 + C1)*(var1 + var2 + C2));
	return ssim;
}
//count avg value of given rectangle 
__host__  __device__ double countAvg(unsigned char * data) {
	double avg = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		avg += data[i];
	}
	avg = avg / (double)RECT_SIZE;
	return avg;
}

//count variance of given rectangle
__host__ __device__ double countVariance(unsigned char * data, double avg) {
	double var = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		var += (data[i] - avg)*(data[i] - avg);
	}
	var = var / (double)RECT_SIZE;
	return var;
}

//count covariance of given rectangle
__host__ __device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2) {
	double cov = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		cov += (data1[i] - avg1)*(data2[i] - avg2);
	}
	cov = cov / (double)RECT_SIZE;
	//if (cov < 0) cout << "neg "<<cov << endl;
	return cov;
}




//count average SSIM value from SSIM values per rectangle
double countRes(double * tmpRes, int count) {
	double sum = 0;
	for (int i = 0; i < count; i += 1) {
		//cout << tmpRes[i]<<endl;
		sum += tmpRes[i];

	}
	//cout << "frame" << endl;
	return sum / (double)count;

}
__global__ void countRectangleKernel(unsigned char * data1, unsigned char * data2, unsigned char * rects1, unsigned char * rects2, double * out, int size, int width) {
	int i = threadIdx.x;
	int j = blockIdx.x;
	int a;
	int pos = j*THREADS + i;
	int x = (pos*SKIP_SIZE) % (width - RECT_SQRT);
	int y = (pos*SKIP_SIZE) / (width - RECT_SQRT)*SKIP_SIZE;
	if (pos<size / SKIP_SIZE / SKIP_SIZE) {
		getRect(data1, x+y*width, width, rects1 + (pos)*RECT_SIZE);
		getRect(data2, x + y*width, width, rects2 + (pos)*RECT_SIZE);
		//return -3;

		out[pos] =  countRectangle(rects1 + pos*RECT_SIZE, rects2 + pos*RECT_SIZE);
	}
}
/*
__global__ void SSIMKernel(unsigned char * data1, unsigned char * data2, double * out, int size, int width){
//
int i = threadIdx.x;
//out[i] =  countSSIM(data1 + size*threadIdx.x, data2 + size*threadIdx.x, size, width);
//countRes(0,0);
}*/



double ** countCUDA(FILE ** streams, FILE * ref, int files_count, PictureData * frame, string type, double ** results) {

	string reference;
	string file1, file2;
	hipError_t cudaStatus;
	
	const int MAX_BUFFER = 2048000;


	//frame2->data = new char[frame->width*frame->height * 3];
	//cout << frame->frame_count << endl;
	
	//cout << frame->width*frame->height / SKIP_SIZE / SKIP_SIZE*RECT_SIZE << endl;
	//hipMalloc((void**)&results, frame2->frame_count*sizeof(double));
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return NULL;
		//goto Error;
	}

	unsigned char ** data=new unsigned char *[files_count];
	unsigned char * dataRef;
	unsigned char ** rects=new unsigned char *[files_count];
	unsigned char ** dataTmp = new unsigned char*[files_count];
	unsigned char * dataTmpTrash = new unsigned char[frame->width*frame->height / 2];
	unsigned char * dataTmpRef = new unsigned char[frame->width*frame->height];
	size_t pitch;

	cudaStatus = hipMalloc((void **)&dataRef, frame->width*frame->height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 1!");
		return NULL;
	}


	//allocated the device memory for source array  
	for (int i = 0; i < files_count; i++) {

		dataTmp[i]= new unsigned char [frame->width*frame->height];
		cudaStatus = hipMalloc((void **)&data[i], frame->width*frame->height);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed 1!");
			return NULL;
		}
	}
	for (int i = 0; i < 2; i++) { //if replaced with files_count it can start kernels in parallel
		cudaStatus = hipMallocPitch((void **)&rects[i], &pitch, RECT_SIZE, frame->width*frame->height / SKIP_SIZE / SKIP_SIZE);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed 3!");
			return NULL;
		}
	}

	
	double * resultsFrame;

	cudaStatus = hipMallocPitch((void**)&resultsFrame, &pitch, sizeof(double), frame->width*frame->height / SKIP_SIZE / SKIP_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 5!");

		return NULL;

	}

	int rec;

	
	for (int i = 0; i < frame->frame_count; i++) {

		rec = fread(dataTmpRef, 1, frame->width*frame->height, ref);
		//fseek(ref, frame->width*frame->height / 2, SEEK_CUR);
		fread(dataTmpTrash, 1, frame->width*frame->height/2, ref); //skip others except Y channel
		if (rec != frame->width*frame->height) {
			printf("error in reading from file 2\n");
			return NULL;
		}
		for (int j = 0; j < files_count; j++) {
			rec = fread(dataTmp[j], 1, frame->width*frame->height, streams[j]);
			if (rec != frame->width*frame->height) {
				printf("error in reading from file 1\n");
				return NULL;
			}
			fread(dataTmpTrash, 1, frame->width*frame->height/2, streams[j]);
			//fseek(streams[j], frame->width*frame->height / 2, SEEK_CUR);//skip others except Y channel

			results[j][i] = countSSIM(dataTmpRef, dataTmp[j], dataRef,data[j], rects[0], rects[1], frame->width*frame->height, frame->width, resultsFrame);
		}
	}

	/*for (int j = 0; j < frame2->frame_count % CHUNK_SIZE; j++) {
	int rec1 = fread(data1[j], 1, frame->width*frame->height * 3, stream);
	int rec2 = fread(data2[j], 1, frame->width*frame->height * 3, stream2);
	if (string(type) == string("SSIM"))  results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
	else results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countPSNR(data1[j], data2[j], frame->width*frame->height);
	//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
	}*/



	//}
	//double * results2=new double[frame2->frame_count];
	/*cudaStatus = hipMemcpy(results2, results, frame2->frame_count, hipMemcpyDeviceToHost);*/
	return results;
}


