#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include "main.h"   
#include "..\..\Video_comparsion\Video_comparsion\PSNR.h"   
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
using namespace std;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__device__ double countAvg(unsigned char * data);
__device__ double countVariance(unsigned char * data, double avg);
__device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2);
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out);
__device__ double countRectangle(unsigned char * data1, unsigned char * data2);
__device__ double countRes(double * tmpRes, int count);
__device__ void getLuma(unsigned char *in, unsigned char *out, int size);

__device__ double countSSIM(unsigned char * datain1, unsigned char * datain2, int size, int width) {
	//unsigned char * data1 = (unsigned char*)datain1;
	//unsigned char * data2 = (unsigned char*)datain2;
	
	float * tmpRes = new float[size/SKIP_SIZE/SKIP_SIZE];
	
	/*unsigned char * data1 = new unsigned char[10];
	unsigned char * data2 = new unsigned char[10];*/
	//getLuma(datain1, data1, size);
	//getLuma(datain2, data2, size);
	unsigned char * rect1 = new unsigned char[RECT_SIZE];
	unsigned char * rect2 = new unsigned char[RECT_SIZE];
	int k = 0;
	
	
	//#pragma omp parallel
	//nthreads = omp_get_num_threads();
	//datain1[10] = 10;
	//	#pragma omp parallel for schedule(static, 20)
	for (int i = 0; i < size / width - RECT_SQRT; i += SKIP_SIZE) {
		for (int j = 0; j < width - RECT_SQRT; j += SKIP_SIZE, k++) {
			//for (int i = 0; i < size-(RECT_SQRT-1)*width; i+=SKIP_SIZE) {
			//data1[0] = 10;
			getRect(datain1, i*width + j, width, rect1);
			getRect(datain2, i*width + j, width, rect2);
			//return -3;
			//tmpRes[k] = 
			//tmpRes[10] = 10;
			/*tmpRes[k] = */countRectangle(rect1, rect2);
			//if (tmpRes[k] < 0) cout << "low result: " << i<< ": " << j<< " :" << tmpRes[k] << endl;
			//delete[] rect1;
			//delete[] rect1;
		}
	}
	
	//double res = countRes(tmpRes, k);
	delete tmpRes;
	/*delete data1;
	delete data2;*/
	delete rect1;
	delete rect2;
	//return res;
}



using namespace std;
__device__ void getLuma(unsigned char *in, unsigned char *out, int size) {
	//#pragma omp parallel for schedule(static, 100)
	for (int i = 0; i < size; i++) {
		//out[i] = round(0.216*in[3 * i] + 0.7152*in[3 * i + 1] + 0.0722*in[3 * i + 2]); //get Luma from RGB picture
		//out[i] = round((double)1/3*in[3 * i] + (double)1/3*in[3 * i + 1] + (double)1/3*in[3 * i + 2]); //get Luma from RGB picture
		out[i] = round(0.299*in[3 * i] + 0.587*in[3 * i + 1] + 0.114*in[3 * i + 2]); //get Luma from RGB picture
	}
}




//return one rectangle with RECT_SIZE pixels
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out) {

	for (int i = 0; i<RECT_SQRT; i++) {
		for (int j = 0; j < RECT_SQRT; j++) {
			out[i*RECT_SQRT + j] = data[start + i*width + j];
		}
		//hipMemcpy(out + i*RECT_SQRT, data + start + i*width, RECT_SQRT, hipMemcpyDeviceToDevice);
	}
	//return out;
}

//count ssim of one rectangle with RECT_SIZE pixels
__device__ double countRectangle(unsigned char * data1, unsigned char * data2) {

	/*double avg1 = countAvg(data1);
	double avg2 = countAvg(data2);*/
	double avg1 = 1;
	double avg2 = 1;
	double var1 = countVariance(data1, avg1);
	double var2 = countVariance(data2, avg2);
	/*
	double cov = countCovariance(data1, data2, avg1, avg2);*/
	
	
	//double ssim = ((2 * avg1*avg2 + C1)*(2 * cov + C2)) / ((avg1*avg1 + avg2*avg2 + C1)*(var1 + var2 + C2));
	return 0;
}
//count avg value of given rectangle 
__device__ double countAvg(unsigned char * data) {
	double avg = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		avg += data[i];
	}
	avg = avg / (double)RECT_SIZE;
	return avg;
}

//count variance of given rectangle
__device__ double countVariance(unsigned char * data, double avg) {
	double var = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		var += (data[i] - avg)*(data[i] - avg);
	}
	var = var / (double)RECT_SIZE;
	return var;
}

//count covariance of given rectangle
__device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2) {
	double cov = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		cov += (data1[i] - avg1)*(data2[i] - avg2);
	}
	cov = cov / (double)RECT_SIZE;
	//if (cov < 0) cout << "neg "<<cov << endl;
	return cov;
}




//count average SSIM value from SSIM values per rectangle
__device__ double countRes(double * tmpRes, int count) {
	double sum = 0;
	for (int i = 0; i < count; i += 1) {
		sum += tmpRes[i];

	}
	return sum / (double)count;

}


__global__ void SSIMKernel(unsigned char * data1, unsigned char * data2, float * out, int size, int width){
	//
	int i = threadIdx.x;
	out[i] =  countSSIM(data1 + size*threadIdx.x, data2 + size*threadIdx.x, size, width);
	//out[10] = 156;
	//countRes(0,0);
}


int compare(const void * a, const void * b)
{
	return (*(double*)a - *(double*)b);
}
PictureData *getVideoInfo(string path) {
	PictureData * data = new PictureData;
	cout << path.c_str() << endl;
	string cmd = "ffprobe -v error -of flat=s=_  -select_streams v:0 -show_entries stream=width,height,r_frame_rate -show_entries format=duration,nb_frames -of default=noprint_wrappers=1:nokey=1 " + path;
	//string cmd="ffprobe -v error -of flat=s=_ -select_streams v:0 -show_entries stream=width,height,nb_frames -of default=noprint_wrappers=1:nokey=1 "+path;
	string cmd2 = "ffprobe - select_streams v - show_streams" + path + " 2> NUL";

#ifdef LINUX
	FILE *stream = popen(cmd.c_str(), "r");
#else 
	FILE *stream = _popen(cmd.c_str(), "r");
#endif
	char buffer[50];
	fgets(buffer, 10, stream);
	data->width = atoi(buffer);
	fgets(buffer, 10, stream);
	data->height = atoi(buffer);
	fgets(buffer, 20, stream);
	string tmp = buffer;
	int pos = tmp.find('/');
	int fps1 = atoi(buffer);
	double fps2 = atoi(tmp.substr(pos + 1).c_str());
	double fps = fps1 / fps2;
	cout << fps << endl;
	fgets(buffer, 20, stream);
	//cout << buffer << endl;


	double len = atof(buffer);

	cout << len*fps << endl;
	data->frame_count = len*fps;
	data->size = data->width*data->height;
	//else data->frame_count = 3121;//181250; // 7100;//3121;//1359;//7192;
	return data;
}
FILE * startFFmpeg(string path) {
#ifdef LINUX
	string cmd = "ffmpeg -i " + path + " -f image2pipe -pix_fmt rgb24 -vcodec rawvideo - 2>/dev/null";
	cout << cmd << endl;
	FILE *stream = popen(cmd.c_str(), "rb");
#else 
	string cmd = "ffmpeg -i " + path + " -f image2pipe -threads 3  -pix_fmt rgb24 -vcodec rawvideo - 2>NUL";
	//-c:v h264_qsv
	FILE *stream = _popen(cmd.c_str(), "rb");
#endif
	cout << cmd.c_str() << endl;


	return stream;
}



int main(int argc, char ** argv){
	string reference;
	string file1, file2;
	string type;
	hipError_t cudaStatus;
	/*if (argc < 6) { // Check the value of argc. If not enough parameters have been passed, inform user and exit.
		cout << argc << endl;
		cout << "Usage is -r <reference file> -in1 <first video to compare> -in2 <second video to compare> [-type]\n"; // Inform the user of how to use the program
																													   //std::cin.get();
		exit(0);
	}
	else { // if we got enough parameters...

		std::cout << argv[0];
		for (int i = 1; i < argc; i++) { 
			if (i + 1 != argc) // Check that we haven't finished parsing already
				if (string(argv[i]) == string("-r")) {
					// We know the next argument *should* be the filename:
					reference = argv[i + 1];
					//std::cout << reference << endl;
				}
				else if (string(argv[i]) == string("-in1")) {
					file1 = string(argv[i + 1]);
					cout << file1.c_str() << endl;
				}
				else if (string(argv[i]) == string("-in2")) {
					file2 = string(argv[i + 1]);
				}
				else if (string(argv[i]) == string("-type")) {
					type = string(argv[i + 1]);

				}
				else {
					//cout << "Not enough or invalid arguments, please try again.\n";
					//Sleep(2000);
					//exit(0);
				}
				std::cout << argv[i] << " ";
		}
	}*/
	file2 = "d:\\Dokumenty\\DP\\dataA\\h265_fHD_5000Kb.mkv";
	file1 = "d:\\Dokumenty\\DP\\dataA\\h264_fHD_5000Kb.mkv";
	type = "SSIM";
	const int MAX_BUFFER = 2048000;

	PictureData * frame = getVideoInfo(file1);
	frame->data = new char[frame->width*frame->height * 3];

	PictureData * frame2 = getVideoInfo(file2);
	frame2->data = new char[frame->width*frame->height * 3];
	cout << frame->frame_count << endl;
	float * results; 
	hipMalloc((void**)&results, frame2->frame_count*sizeof(float));
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
		//goto Error;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		fprintf(stderr, (char *)cudaStatus);
		return -1;
	}
	FILE * stream = startFFmpeg(file1);
	FILE * stream2 = startFFmpeg(file2);

	unsigned char * data1;
	unsigned char * data2;
	unsigned char * data3;
	unsigned char * data4;

	size_t pitch;
	//allocated the device memory for source array  
	cudaStatus=hipMallocPitch((void **)&data2, &pitch, frame->width*frame->height * 3, CHUNK_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMallocPitch((void **)&data1, &pitch, frame->width*frame->height * 3, CHUNK_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	/*hipMalloc((void**)&data1, CHUNK_SIZE*sizeof(unsigned char *));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	hipMalloc((void**)&data2, CHUNK_SIZE * sizeof(unsigned char *));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	for (int j = 0; j < CHUNK_SIZE; j++) {
		hipMalloc((void**)&data1[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		hipMalloc((void**)&data2[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		


		/*data1[j] = new unsigned char[frame->width*frame->height * 3];
		data2[j] = new unsigned char[frame->width*frame->height * 3];
	}*/
	unsigned char * datatmp;
	datatmp = new unsigned char[frame->width*frame->height * 3];
	for (int i = 0; i < frame2->frame_count / CHUNK_SIZE; i++) {
		for (int j = 0; j < CHUNK_SIZE; j++) {
			int rec1 = fread(datatmp, 1, frame->width*frame->height * 3, stream);
			cudaStatus = hipMemcpy(data1,datatmp,frame->width*frame->height * 3, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				printf("%s\n", hipGetErrorString(cudaStatus));
				return -1;
			}


			if (rec1 != frame->width*frame->height * 3) {
				printf("error while reading file 1\n");
				return -1;
			}

			int rec2 = fread(datatmp, 1, frame->width*frame->height * 3, stream2);
			cudaStatus = hipMemcpy(data2, datatmp, frame->width*frame->height * 3, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				if (cudaStatus != hipSuccess) printf("%s\n", hipGetErrorString(cudaStatus));
				return -1;
			}

			if (rec2 != frame->width*frame->height * 3) {
				cout << "error2" << endl;
				return -1;
			}
		}
		//SSIMKernel(unsigned char ** data1, unsigned char ** data2, float * data3, float * data4, int size, int width)
		SSIMKernel<<<1, 1 >>>(data1, data2, results, frame->size, frame->width);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			if (cudaStatus != hipSuccess) printf("%s\n", hipGetErrorString(cudaStatus));
			return -1;
		}

		float * resultsHost = new float[frame2->frame_count];
		cudaStatus = hipMemcpy(resultsHost, results, frame->frame_count * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			if (cudaStatus != hipSuccess) printf("%s\n", hipGetErrorString(cudaStatus));
			return -1;
		}
		
		
		cout << resultsHost[10] << endl;
		/*
		omp_set_num_threads(CHUNK_SIZE);
#pragma omp parallel for 
		for (int j = 0; j < CHUNK_SIZE; j++) {
			if (string(type) == string("SSIM")) results[j + i*CHUNK_SIZE] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
			else results[j + i*CHUNK_SIZE] = countPSNR(data1[j], data2[j], frame->width*frame->height);
			//cout << j+i * CHUNK_SIZE << " " << results[j+i*CHUNK_SIZE] << endl;
		}*/
	}

	/*for (int j = 0; j < frame2->frame_count % CHUNK_SIZE; j++) {
		int rec1 = fread(data1[j], 1, frame->width*frame->height * 3, stream);
		int rec2 = fread(data2[j], 1, frame->width*frame->height * 3, stream2);
		if (string(type) == string("SSIM"))  results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
		else results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countPSNR(data1[j], data2[j], frame->width*frame->height);
		//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
	}*/



	//}
	float * results2=new float[frame2->frame_count];
	cudaStatus = hipMemcpy(results2, results, frame2->frame_count, hipMemcpyDeviceToHost);
	double sum = 0;
	int frames = frame2->frame_count;
	for (int i = 0; i<frame2->frame_count; i++) {
		cout << i << " " << results2[i] << endl;
		if (isfinite(results2[i]))
			sum += results2[i];
		else frames--;
	}

	delete frame->data;
	delete frame2->data;
	delete frame;
	delete frame2;
	cout << "AVG: " << sum / frames << endl;
	qsort(results2, frames, sizeof(double), compare);
	cout << "Median: " << results2[frames / 2] << endl;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
