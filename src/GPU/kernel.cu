#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include "main.h"   
//#include "..\..\Video_comparsion\Video_comparsion\PSNR.h"   
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <errno.h>
#include <stdio.h>
using namespace std;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__device__ float countAvg(unsigned char * data);
__device__ float countVariance(unsigned char * data, float avg);
__device__ float countCovariance(unsigned char * data1, unsigned char * data2, float avg1, float avg2);
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out);
__device__ float countRectangle(unsigned char * data1, unsigned char * data2);
float countRes(float * tmpRes, int count);
void getLuma(unsigned char *in, unsigned char *out, int size);

float countSSIM(unsigned char * datain1, unsigned char * datain2,unsigned char * dataC1, unsigned char * dataC2, unsigned char * rects1,unsigned char * rects2,int size, int width,float*& results) {
	//unsigned char * data1 = (unsigned char*)datain1;
	//unsigned char * data2 = (unsigned char*)datain2;
	hipError_t cudaStatus;
	//float * tmpRes = new float[size];
	
	unsigned char * data1 = new unsigned char[size];
	unsigned char * data2 = new unsigned char[size];
/*
	if (data1==0 or data2==0){
		//return -1;
		//cout<<"error in allocation"<<endl;
	}*/
	getLuma(datain1, data1, size);
	getLuma(datain2, data2, size);
	/*unsigned char * rect1 = new unsigned char[RECT_SIZE];
	unsigned char * rect2 = new unsigned char[RECT_SIZE];
	int k = 0;*/

	
	
	//#pragma omp parallel
	//nthreads = omp_get_num_threads();

	
	cudaStatus = hipMemcpy(dataC1,data1,size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess ) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	cudaStatus = hipMemcpy(dataC2,data2,size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}


	//	#pragma omp parallel for schedule(static, 20)
	int rectCount=size/SKIP_SIZE/SKIP_SIZE/THREADS*THREADS;
	int blocks=rectCount/THREADS;
	if (rectCount<size/SKIP_SIZE/SKIP_SIZE){
		blocks=rectCount/THREADS+1;
	}
	countRectangleKernel<<<blocks,THREADS>>>(dataC1,dataC2,rects1,rects2,results,size,width); //FIXME - need to adjust size to count up to THREADS last rectangles!!
	hipDeviceSynchronize();
	/*for (int i = 0; i < size / width - RECT_SQRT; i += SKIP_SIZE) {

		for (int j = 0; j < width - RECT_SQRT; j += SKIP_SIZE, k++) {
			//for (int i = 0; i < size-(RECT_SQRT-1)*width; i+=SKIP_SIZE) {

			//if (tmpRes[k] < 0) cout << "low result: " << i<< ": " << j<< " :" << tmpRes[k] << endl;

		}
	}


	float res = countRes(tmpRes, k);
	delete[] tmpRes;
	delete[] data1;
	delete[] data2;
	delete[] rect1;
	delete[] rect2;*/
	
	float * resultsOut=new float[rectCount];
	cudaStatus = hipMemcpy(resultsOut,results,rectCount*sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	float output=countRes(resultsOut,rectCount);
	cout<<output<<endl;
	return output;
}



using namespace std;
void getLuma(unsigned char *in, unsigned char *out, int size) {
	//#pragma omp parallel for schedule(static, 100)
	for (int i = 0; i < size; i++) {
		//out[i] = round(0.216*in[3 * i] + 0.7152*in[3 * i + 1] + 0.0722*in[3 * i + 2]); //get Luma from RGB picture
		//out[i] = round((float)1/3*in[3 * i] + (float)1/3*in[3 * i + 1] + (float)1/3*in[3 * i + 2]); //get Luma from RGB picture
		out[i] = round(0.299*in[3 * i] + 0.587*in[3 * i + 1] + 0.114*in[3 * i + 2]); //get Luma from RGB picture
	}
}




//return one rectangle with RECT_SIZE pixels
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out) {

	for (int i = 0; i<RECT_SQRT; i++) {
		for (int j = 0; j < RECT_SQRT; j++) {
			out[i*RECT_SQRT + j] = data[start + i*width + j];
		}
		//hipMemcpy(out + i*RECT_SQRT, data + start + i*width, RECT_SQRT, hipMemcpyDeviceToDevice);
	}
	//return out;
}

//count ssim of one rectangle with RECT_SIZE pixels
__device__ float countRectangle(unsigned char * data1, unsigned char * data2) {

	float avg1 = countAvg(data1);
	float avg2 = countAvg(data2);

	float var1 = countVariance(data1, avg1);
	float var2 = countVariance(data2, avg2);

	float cov = countCovariance(data1, data2, avg1, avg2);


	float ssim = ((2 * avg1*avg2 + C1)*(2 * cov + C2)) / ((avg1*avg1 + avg2*avg2 + C1)*(var1 + var2 + C2));
	return ssim;
}
//count avg value of given rectangle 
__device__ float countAvg(unsigned char * data) {
	float avg = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		avg += data[i];
	}
	avg = avg / (float)RECT_SIZE;
	return avg;
}

//count variance of given rectangle
__device__ float countVariance(unsigned char * data, float avg) {
	float var = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		var += (data[i] - avg)*(data[i] - avg);
	}
	var = var / (float)RECT_SIZE;
	return var;
}

//count covariance of given rectangle
__device__ float countCovariance(unsigned char * data1, unsigned char * data2, float avg1, float avg2) {
	float cov = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		cov += (data1[i] - avg1)*(data2[i] - avg2);
	}
	cov = cov / (float)RECT_SIZE;
	//if (cov < 0) cout << "neg "<<cov << endl;
	return cov;
}




//count average SSIM value from SSIM values per rectangle
float countRes(float * tmpRes, int count) {
	float sum = 0;
	for (int i = 0; i < count; i += 1) {
		sum += tmpRes[i];

	}
	return sum / (float)count;

}
__global__ void countRectangleKernel(unsigned char * data1, unsigned char * data2,unsigned char * rects1,unsigned char * rects2,float * out,int size, int width){
			int i = threadIdx.x;
			int j= blockIdx.x;
int a;
			int pos=j*THREADS+i;
			if (pos<size/SKIP_SIZE/SKIP_SIZE){
				getRect(data1, ((pos)* SKIP_SIZE)%width + (pos*SKIP_SIZE)/width*SKIP_SIZE*width, width, rects1+(pos)*RECT_SIZE);
				getRect(data2, ((pos)* SKIP_SIZE)%width + (pos*SKIP_SIZE)/width*SKIP_SIZE*width, width, rects2+(pos)*RECT_SIZE);
				//return -3;

				out[pos] = countRectangle(rects1+pos*RECT_SIZE, rects2+pos*RECT_SIZE);
			}
	else{
	rects1[0]=1;
}
a=a+1;
}
/*
__global__ void SSIMKernel(unsigned char * data1, unsigned char * data2, float * out, int size, int width){
	//
	int i = threadIdx.x;
	//out[i] =  countSSIM(data1 + size*threadIdx.x, data2 + size*threadIdx.x, size, width);
	//countRes(0,0);
}*/


int compare(const void * a, const void * b)
{
	return (*(float*)a - *(float*)b);
}
PictureData *getVideoInfo(string path) {
	PictureData * data = new PictureData;
	cout << path.c_str() << endl;
	string cmd = "ffprobe -v error -of flat=s=_  -select_streams v:0 -show_entries stream=width,height,r_frame_rate -show_entries format=duration,nb_frames -of default=noprint_wrappers=1:nokey=1 " + path;
	//string cmd="ffprobe -v error -of flat=s=_ -select_streams v:0 -show_entries stream=width,height,nb_frames -of default=noprint_wrappers=1:nokey=1 "+path;
	string cmd2 = "ffprobe - select_streams v - show_streams" + path + " 2> NUL";

#ifdef __linux__
	FILE *stream = popen(cmd.c_str(), "r");
#else 
	FILE *stream = _popen(cmd.c_str(), "r");
#endif
	char buffer[50];
	fgets(buffer, 10, stream);
	data->width = atoi(buffer);
	fgets(buffer, 10, stream);
	data->height = atoi(buffer);
	fgets(buffer, 20, stream);
	string tmp = buffer;
	int pos = tmp.find('/');
	int fps1 = atoi(buffer);
	float fps2 = atoi(tmp.substr(pos + 1).c_str());
	float fps = fps1 / fps2;
	cout << fps << endl;
	fgets(buffer, 20, stream);
	//cout << buffer << endl;


	float len = atof(buffer);

	cout << len*fps << endl;
	data->frame_count = len*fps;
	//else data->frame_count = 3121;//181250; // 7100;//3121;//1359;//7192;
	return data;
}
void startFFmpeg(string path, FILE *& stream) {
#ifdef __linux__
	string cmd = "ffmpeg -i " + path + " -f image2pipe -pix_fmt rgb24 -vcodec rawvideo - 2>/dev/null";
	cout << cmd << endl;
	stream = popen(cmd.c_str(), "r");
#else 
	string cmd = "ffmpeg -i " + path + " -f image2pipe -threads 3  -pix_fmt rgb24 -vcodec rawvideo - 2>NUL";
	//-c:v h264_qsv
	stream = _popen(cmd.c_str(), "rb");
#endif
	cout << cmd.c_str() << endl;

	if (stream == NULL)
    		printf ("Error opening file: %s\n",strerror(errno));

	//return stream;
}



int main(int argc, char ** argv){
	string reference;
	string file1, file2;
	string type;
	hipError_t cudaStatus;
	if (argc < 6) { // Check the value of argc. If not enough parameters have been passed, inform user and exit.
		cout << argc << endl;
		cout << "Usage is -r <reference file> -in1 <first video to compare> -in2 <second video to compare> [-type]\n"; // Inform the user of how to use the program
																													   //std::cin.get();
		exit(0);
	}
	else { // if we got enough parameters...

		std::cout << argv[0];
		for (int i = 1; i < argc; i++) { /* We will iterate over argv[] to get the parameters stored inside.
										 * Note that we're starting on 1 because we don't need to know the
										 * path of the program, which is stored in argv[0] */
			if (i + 1 != argc) // Check that we haven't finished parsing already
				if (string(argv[i]) == string("-r")) {
					// We know the next argument *should* be the filename:
					reference = argv[i + 1];
					//std::cout << reference << endl;
				}
				else if (string(argv[i]) == string("-in1")) {
					file1 = string(argv[i + 1]);
					cout << file1.c_str() << endl;
				}
				else if (string(argv[i]) == string("-in2")) {
					file2 = string(argv[i + 1]);
				}
				else if (string(argv[i]) == string("-type")) {
					type = string(argv[i + 1]);

				}
				else {
					//cout << "Not enough or invalid arguments, please try again.\n";
					//Sleep(2000);
					//exit(0);
				}
				std::cout << argv[i] << " ";
		}
	}

	const int MAX_BUFFER = 2048000;

	PictureData * frame = getVideoInfo(file1);
	frame->data = new char[frame->width*frame->height * 3];

	PictureData * frame2 = getVideoInfo(file2);
	frame2->data = new char[frame->width*frame->height * 3];
	cout << frame->frame_count << endl;
	float * results; 

	FILE * stream=0;
	startFFmpeg(file1, stream) ;

	FILE * stream2=0;
	startFFmpeg(file2, stream2) ;
	if (stream == NULL)
    		printf ("Error opening file: %s\n",strerror(errno));
	if (stream2 == NULL)
    		printf ("Error opening file2: %s\n",strerror(errno));

	results=new float[frame2->frame_count];	
//hipMalloc((void**)&results, frame2->frame_count*sizeof(float));
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
		//goto Error;
	}

	unsigned char * data1;
	unsigned char * data2;
	unsigned char * rects1;
	unsigned char * rects2;
	size_t pitch;
	//allocated the device memory for source array  
	cudaStatus=hipMalloc((void **)&data2, frame->width*frame->height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMalloc((void **)&data1, frame->width*frame->height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMallocPitch((void **)&rects1,&pitch, RECT_SIZE,frame->width*frame->height/SKIP_SIZE/SKIP_SIZE*RECT_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMallocPitch((void **)&rects2,&pitch, RECT_SIZE,frame->width*frame->height/SKIP_SIZE/SKIP_SIZE*RECT_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	
	float * resultsFrame;
	cudaStatus=hipMalloc((void**)&resultsFrame, frame->width*frame->height/SKIP_SIZE/SKIP_SIZE*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

		return -1;

	}

	/*for (int j = 0; j < CHUNK_SIZE; j++) {
		hipMalloc((void**)&data1[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		hipMalloc((void**)&data2[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		
*/

		/*data1[j] = new unsigned char[frame->width*frame->height * 3];
		data2[j] = new unsigned char[frame->width*frame->height * 3];
	}*/
	unsigned char * datatmp1 = new unsigned char[frame->width*frame->height * 3];
	unsigned char * datatmp2 = new unsigned char[frame->width*frame->height * 3];
	for (int i = 0; i < frame2->frame_count ; i++) {

		int rec1 = fread(datatmp1, 1, frame->width*frame->height * 3, stream);
		if (rec1 != frame->width*frame->height*3) {
			printf("error in reading from file 1\n");
			return -1;
		}		


		int rec2 = fread(datatmp2, 1, frame->width*frame->height * 3, stream2);
		if (rec2 != frame->width*frame->height*3) {
			printf("error in reading from file 2\n");
			return -1;
		}


		//float countSSIM(unsigned char * datain1, unsigned char * datain2,unsigned char * dataC1, unsigned char * dataC2, unsigned char ** rects1,unsigned char ** rects2,int size, int width
		results[i]=countSSIM(datatmp1,datatmp2,data1, data2, rects1,rects2, frame->width*frame->height, frame->width,resultsFrame);
		/*
		omp_set_num_threads(CHUNK_SIZE);
#pragma omp parallel for 
		for (int j = 0; j < CHUNK_SIZE; j++) {
			if (string(type) == string("SSIM")) results[j + i*CHUNK_SIZE] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
			else results[j + i*CHUNK_SIZE] = countPSNR(data1[j], data2[j], frame->width*frame->height);
			//cout << j+i * CHUNK_SIZE << " " << results[j+i*CHUNK_SIZE] << endl;
		}*/
	}

	/*for (int j = 0; j < frame2->frame_count % CHUNK_SIZE; j++) {
		int rec1 = fread(data1[j], 1, frame->width*frame->height * 3, stream);
		int rec2 = fread(data2[j], 1, frame->width*frame->height * 3, stream2);
		if (string(type) == string("SSIM"))  results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
		else results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countPSNR(data1[j], data2[j], frame->width*frame->height);
		//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
	}*/



	//}
	//float * results2=new float[frame2->frame_count];
	/*cudaStatus = hipMemcpy(results2, results, frame2->frame_count, hipMemcpyDeviceToHost);*/
	float sum = 0;
	int frames = frame2->frame_count;
	for (int i = 0; i<frame2->frame_count; i++) {
		cout << i << " " << results[i] << endl;
		if (isfinite(results[i]))
			sum += results[i];
		else frames--;
	}

	cout << "AVG: " << sum / frames << endl;
	qsort(results, frames, sizeof(float), compare);
	cout << "Median: " << results[frames / 2] << endl;
}


