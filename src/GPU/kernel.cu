#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include "main.h"   
//#include "..\..\Video_comparsion\Video_comparsion\PSNR.h"   
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <errno.h>
#include <stdio.h>
using namespace std;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__device__ double countAvg(unsigned char * data);
__device__ double countVariance(unsigned char * data, double avg);
__device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2);
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out);
__device__ double countRectangle(unsigned char * data1, unsigned char * data2);
float countRes(float * tmpRes, int count);
void getLuma(unsigned char *in, unsigned char *out, int size);

double countSSIM(unsigned char * datain1, unsigned char * datain2,unsigned char * dataC1, unsigned char * dataC2, unsigned char ** rects1,unsigned char ** rects2,int size, int width) {
	//unsigned char * data1 = (unsigned char*)datain1;
	//unsigned char * data2 = (unsigned char*)datain2;
	hipError_t cudaStatus;
	//double * tmpRes = new double[size];
	
	unsigned char * data1 = new unsigned char[size];
	unsigned char * data2 = new unsigned char[size];
/*
	if (data1==0 or data2==0){
		//return -1;
		//cout<<"error in allocation"<<endl;
	}*/
	getLuma(datain1, data1, size);
	getLuma(datain2, data2, size);
	/*unsigned char * rect1 = new unsigned char[RECT_SIZE];
	unsigned char * rect2 = new unsigned char[RECT_SIZE];
	int k = 0;*/

	float * results;
	cudaStatus=hipMalloc((void**)&results, size/SKIP_SIZE/SKIP_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

		return -1;

	}
	//#pragma omp parallel
	//nthreads = omp_get_num_threads();

	
	cudaStatus = hipMemcpy(dataC1,data1,size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess ) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	cudaStatus = hipMemcpy(dataC2,data2,size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}


	//	#pragma omp parallel for schedule(static, 20)
	countRectangleKernel<<<size/SKIP_SIZE/SKIP_SIZE/THREADS,THREADS>>>(dataC1,dataC2,rects1,rects2,results,size,width); //FIXME - need to adjust size to count up to THREADS last rectangles!!
	/*for (int i = 0; i < size / width - RECT_SQRT; i += SKIP_SIZE) {

		for (int j = 0; j < width - RECT_SQRT; j += SKIP_SIZE, k++) {
			//for (int i = 0; i < size-(RECT_SQRT-1)*width; i+=SKIP_SIZE) {

			//if (tmpRes[k] < 0) cout << "low result: " << i<< ": " << j<< " :" << tmpRes[k] << endl;

		}
	}


	double res = countRes(tmpRes, k);
	delete[] tmpRes;
	delete[] data1;
	delete[] data2;
	delete[] rect1;
	delete[] rect2;*/

	float * resultsOut=new float[size/SKIP_SIZE/SKIP_SIZE];
	cudaStatus = hipMemcpy(resultsOut,results,size/SKIP_SIZE/SKIP_SIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("%s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	double output=countRes(resultsOut, size/SKIP_SIZE/SKIP_SIZE);
	return output;
}



using namespace std;
void getLuma(unsigned char *in, unsigned char *out, int size) {
	//#pragma omp parallel for schedule(static, 100)
	for (int i = 0; i < size; i++) {
		//out[i] = round(0.216*in[3 * i] + 0.7152*in[3 * i + 1] + 0.0722*in[3 * i + 2]); //get Luma from RGB picture
		//out[i] = round((double)1/3*in[3 * i] + (double)1/3*in[3 * i + 1] + (double)1/3*in[3 * i + 2]); //get Luma from RGB picture
		out[i] = round(0.299*in[3 * i] + 0.587*in[3 * i + 1] + 0.114*in[3 * i + 2]); //get Luma from RGB picture
	}
}




//return one rectangle with RECT_SIZE pixels
__device__ void getRect(unsigned char* data, int start, int width, unsigned char * out) {

	for (int i = 0; i<RECT_SQRT; i++) {
		for (int j = 0; j < RECT_SQRT; j++) {
			out[i*RECT_SQRT + j] = data[start + i*width + j];
		}
		//hipMemcpy(out + i*RECT_SQRT, data + start + i*width, RECT_SQRT, hipMemcpyDeviceToDevice);
	}
	//return out;
}

//count ssim of one rectangle with RECT_SIZE pixels
__device__ double countRectangle(unsigned char * data1, unsigned char * data2) {

	double avg1 = countAvg(data1);
	double avg2 = countAvg(data2);

	double var1 = countVariance(data1, avg1);
	double var2 = countVariance(data2, avg2);

	double cov = countCovariance(data1, data2, avg1, avg2);


	double ssim = ((2 * avg1*avg2 + C1)*(2 * cov + C2)) / ((avg1*avg1 + avg2*avg2 + C1)*(var1 + var2 + C2));
	return ssim;
}
//count avg value of given rectangle 
__device__ double countAvg(unsigned char * data) {
	double avg = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		avg += data[i];
	}
	avg = avg / (double)RECT_SIZE;
	return avg;
}

//count variance of given rectangle
__device__ double countVariance(unsigned char * data, double avg) {
	double var = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		var += (data[i] - avg)*(data[i] - avg);
	}
	var = var / (double)RECT_SIZE;
	return var;
}

//count covariance of given rectangle
__device__ double countCovariance(unsigned char * data1, unsigned char * data2, double avg1, double avg2) {
	double cov = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		cov += (data1[i] - avg1)*(data2[i] - avg2);
	}
	cov = cov / (double)RECT_SIZE;
	//if (cov < 0) cout << "neg "<<cov << endl;
	return cov;
}




//count average SSIM value from SSIM values per rectangle
float countRes(float * tmpRes, int count) {
	float sum = 0;
	for (int i = 0; i < count; i += 1) {
		sum += tmpRes[i];

	}
	return sum / (float)count;

}
__global__ void countRectangleKernel(unsigned char * data1, unsigned char * data2,unsigned char **rects1,unsigned char ** rects2,float * out,int size, int width){
			int i = threadIdx.x;
			int j= blockIdx.x;
			getRect(data1, (j*THREADS+i)* SKIP_SIZE * SKIP_SIZE, width, rects1[i]);
			getRect(data2, (j*THREADS+i)* SKIP_SIZE* SKIP_SIZE, width, rects2[i]);
			//return -3;

			out[i] = countRectangle(rects1[i], rects2[i]);
}
/*
__global__ void SSIMKernel(unsigned char * data1, unsigned char * data2, float * out, int size, int width){
	//
	int i = threadIdx.x;
	//out[i] =  countSSIM(data1 + size*threadIdx.x, data2 + size*threadIdx.x, size, width);
	//countRes(0,0);
}*/


int compare(const void * a, const void * b)
{
	return (*(double*)a - *(double*)b);
}
PictureData *getVideoInfo(string path) {
	PictureData * data = new PictureData;
	cout << path.c_str() << endl;
	string cmd = "ffprobe -v error -of flat=s=_  -select_streams v:0 -show_entries stream=width,height,r_frame_rate -show_entries format=duration,nb_frames -of default=noprint_wrappers=1:nokey=1 " + path;
	//string cmd="ffprobe -v error -of flat=s=_ -select_streams v:0 -show_entries stream=width,height,nb_frames -of default=noprint_wrappers=1:nokey=1 "+path;
	string cmd2 = "ffprobe - select_streams v - show_streams" + path + " 2> NUL";

#ifdef __linux__
	FILE *stream = popen(cmd.c_str(), "r");
#else 
	FILE *stream = _popen(cmd.c_str(), "r");
#endif
	char buffer[50];
	fgets(buffer, 10, stream);
	data->width = atoi(buffer);
	fgets(buffer, 10, stream);
	data->height = atoi(buffer);
	fgets(buffer, 20, stream);
	string tmp = buffer;
	int pos = tmp.find('/');
	int fps1 = atoi(buffer);
	double fps2 = atoi(tmp.substr(pos + 1).c_str());
	double fps = fps1 / fps2;
	cout << fps << endl;
	fgets(buffer, 20, stream);
	//cout << buffer << endl;


	double len = atof(buffer);

	cout << len*fps << endl;
	data->frame_count = len*fps;
	//else data->frame_count = 3121;//181250; // 7100;//3121;//1359;//7192;
	return data;
}
void startFFmpeg(string path, FILE *& stream) {
#ifdef __linux__
	string cmd = "ffmpeg -i " + path + " -f image2pipe -pix_fmt rgb24 -vcodec rawvideo - 2>/dev/null";
	cout << cmd << endl;
	stream = popen(cmd.c_str(), "r");
#else 
	string cmd = "ffmpeg -i " + path + " -f image2pipe -threads 3  -pix_fmt rgb24 -vcodec rawvideo - 2>NUL";
	//-c:v h264_qsv
	stream = _popen(cmd.c_str(), "rb");
#endif
	cout << cmd.c_str() << endl;

	if (stream == NULL)
    		printf ("Error opening file: %s\n",strerror(errno));

	//return stream;
}



int main(int argc, char ** argv){
	string reference;
	string file1, file2;
	string type;
	hipError_t cudaStatus;
	if (argc < 6) { // Check the value of argc. If not enough parameters have been passed, inform user and exit.
		cout << argc << endl;
		cout << "Usage is -r <reference file> -in1 <first video to compare> -in2 <second video to compare> [-type]\n"; // Inform the user of how to use the program
																													   //std::cin.get();
		exit(0);
	}
	else { // if we got enough parameters...

		std::cout << argv[0];
		for (int i = 1; i < argc; i++) { /* We will iterate over argv[] to get the parameters stored inside.
										 * Note that we're starting on 1 because we don't need to know the
										 * path of the program, which is stored in argv[0] */
			if (i + 1 != argc) // Check that we haven't finished parsing already
				if (string(argv[i]) == string("-r")) {
					// We know the next argument *should* be the filename:
					reference = argv[i + 1];
					//std::cout << reference << endl;
				}
				else if (string(argv[i]) == string("-in1")) {
					file1 = string(argv[i + 1]);
					cout << file1.c_str() << endl;
				}
				else if (string(argv[i]) == string("-in2")) {
					file2 = string(argv[i + 1]);
				}
				else if (string(argv[i]) == string("-type")) {
					type = string(argv[i + 1]);

				}
				else {
					//cout << "Not enough or invalid arguments, please try again.\n";
					//Sleep(2000);
					//exit(0);
				}
				std::cout << argv[i] << " ";
		}
	}

	const int MAX_BUFFER = 2048000;

	PictureData * frame = getVideoInfo(file1);
	frame->data = new char[frame->width*frame->height * 3];

	PictureData * frame2 = getVideoInfo(file2);
	frame2->data = new char[frame->width*frame->height * 3];
	cout << frame->frame_count << endl;
	float * results; 

	FILE * stream=0;
	startFFmpeg(file1, stream) ;

	FILE * stream2=0;
	startFFmpeg(file2, stream2) ;
	if (stream == NULL)
    		printf ("Error opening file: %s\n",strerror(errno));
	if (stream2 == NULL)
    		printf ("Error opening file2: %s\n",strerror(errno));

	results=new float[frame2->frame_count];	
//hipMalloc((void**)&results, frame2->frame_count*sizeof(float));
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
		//goto Error;
	}

	unsigned char * data1;
	unsigned char * data2;
	unsigned char ** rects1;
	unsigned char ** rects2;
	size_t pitch;
	//allocated the device memory for source array  
	cudaStatus=hipMalloc((void **)&data2, frame->width*frame->height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMalloc((void **)&data1, frame->width*frame->height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMallocPitch((void **)&rects1,&pitch, RECT_SIZE,frame->width*frame->height/SKIP_SIZE/SKIP_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}
	cudaStatus=hipMallocPitch((void **)&rects2,&pitch, RECT_SIZE,frame->width*frame->height/SKIP_SIZE/SKIP_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return -1;
	}


	/*for (int j = 0; j < CHUNK_SIZE; j++) {
		hipMalloc((void**)&data1[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		hipMalloc((void**)&data2[j], frame->width*frame->height * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return -1;
		}
		
*/

		/*data1[j] = new unsigned char[frame->width*frame->height * 3];
		data2[j] = new unsigned char[frame->width*frame->height * 3];
	}*/
	unsigned char * datatmp1 = new unsigned char[frame->width*frame->height * 3];
	unsigned char * datatmp2 = new unsigned char[frame->width*frame->height * 3];
	for (int i = 0; i < frame2->frame_count ; i++) {

		int rec1 = fread(datatmp1, 1, frame->width*frame->height * 3, stream);
		if (rec1 != frame->width*frame->height*3) {
			printf("error in reading from file 1\n");
			return -1;
		}		


		int rec2 = fread(datatmp2, 1, frame->width*frame->height * 3, stream2);
		if (rec2 != frame->width*frame->height*3) {
			printf("error in reading from file 2\n");
			return -1;
		}


		//double countSSIM(unsigned char * datain1, unsigned char * datain2,unsigned char * dataC1, unsigned char * dataC2, unsigned char ** rects1,unsigned char ** rects2,int size, int width
		results[i]=countSSIM(datatmp1,datatmp2,data1, data2, rects1,rects2, frame->width*frame->height, frame->width);
		/*
		omp_set_num_threads(CHUNK_SIZE);
#pragma omp parallel for 
		for (int j = 0; j < CHUNK_SIZE; j++) {
			if (string(type) == string("SSIM")) results[j + i*CHUNK_SIZE] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
			else results[j + i*CHUNK_SIZE] = countPSNR(data1[j], data2[j], frame->width*frame->height);
			//cout << j+i * CHUNK_SIZE << " " << results[j+i*CHUNK_SIZE] << endl;
		}*/
	}

	/*for (int j = 0; j < frame2->frame_count % CHUNK_SIZE; j++) {
		int rec1 = fread(data1[j], 1, frame->width*frame->height * 3, stream);
		int rec2 = fread(data2[j], 1, frame->width*frame->height * 3, stream2);
		if (string(type) == string("SSIM"))  results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countSSIM(data1[j], data2[j], frame->width*frame->height, frame->width);
		else results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] = countPSNR(data1[j], data2[j], frame->width*frame->height);
		//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
	}*/



	//}
	//float * results2=new float[frame2->frame_count];
	/*cudaStatus = hipMemcpy(results2, results, frame2->frame_count, hipMemcpyDeviceToHost);*/
	double sum = 0;
	int frames = frame2->frame_count;
	for (int i = 0; i<frame2->frame_count; i++) {
		cout << i << " " << results[i] << endl;
		if (isfinite(results[i]))
			sum += results[i];
		else frames--;
	}

	cout << "AVG: " << sum / frames << endl;
	qsort(results, frames, sizeof(double), compare);
	cout << "Median: " << results[frames / 2] << endl;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
