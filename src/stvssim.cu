#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "stvssim.cuh"
#include "stvssim.h"
#include "SSIM.h"
#include <stdlib.h>     /* abs */
#include <map>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;


void cudaTest(hipError_t cudaStatus, string descr) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%d", int(cudaStatus));
		fprintf(stderr, strcat("cuda failed: ", descr.c_str()));
		exit(-1);
	}

}

//count STVSSIM metric for all files 
double ** countMetricSTVSSIM_CUDA(FILE ** streams, FILE * ref, int files_count, PictureData * frame, double ** results, int *& frames) {
	hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return NULL;
		//goto Error;
	}
	//results= new double *[files_count];
#define CUDA_RUNNING 1
	int rec;
	
	size_t pitch;

	//unsigned char * ref_dataHost=new unsigned char frame->size* FRAME_CNT;
	unsigned char * ref_data = new unsigned char[FRAME_CNT*frame->size];
	//cudaStatus = hipMallocPitch((void **)&ref_data, &pitch, frame->size, FRAME_CNT);
	//cudaTest(cudaStatus, "malloc 1");

	//unsigned char * dataHost = NULL;
	unsigned char * data  = new unsigned char [FRAME_CNT*frame->size];
	//cudaStatus = hipMallocPitch((void **)&data, &pitch, frame->size, FRAME_CNT*files_count);
	//cudaTest(cudaStatus, "malloc 2");


	unsigned char * dataTrash = new unsigned char[frame->size / 2];
	unsigned char * dataTmp = new unsigned char[frame->size];


	for (int i = FRAME_CNT / 2; i < FRAME_CNT; i++) {
		for (int j = 0; j < files_count; j++) {
			readFromFile(dataTmp, frame->size, streams[j]);
			readFromFile(dataTrash, frame->size / 2, streams[j]);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			memcpy(data + indexFs(i, 0, frame->size, j), dataTmp, frame->size);
			//hipMemcpy(data + indexFs(i, 0, frame->size,j),dataTmp,frame->size, hipMemcpyHostToDevice);
		}
		readFromFile(dataTmp, frame->size, ref);
		readFromFile(dataTrash, frame->size / 2, ref);
		memcpy(ref_data + indexF(i, 0, frame->size), dataTmp, frame->size);
		//hipMemcpy(ref_data + indexF(i, 0, frame->size), dataTmp, frame->size, hipMemcpyHostToDevice);
	}

	int i = FRAME_SKIP;
	int j = 0;
	data_CUDA * dataCuda=new data_CUDA;
	
	cudaStatus = hipMallocPitch((void **)&dataCuda->filters, &pitch, RECT_SIZE_3D, FRAME_CNT*4);//4 different filters with size of one cube
	cudaTest(cudaStatus, "malloc filters");
	cudaStatus = hipMallocPitch((void **)&dataCuda->datain1, &pitch, FRAME_CNT, frame->size);
	cudaTest(cudaStatus, "malloc datain1");
	cudaStatus = hipMallocPitch((void **)&dataCuda->datain2, &pitch, FRAME_CNT, frame->size );
	int k = (frame->size / frame->width - RECT_SQRT) / SKIP_SIZE*(frame->width - RECT_SQRT) / SKIP_SIZE;
	cudaStatus = hipMallocPitch((void **)&dataCuda->tmpRes, &pitch, sizeof(double), k);
	cudaTest(cudaStatus, "malloc datain2");
	cudaStatus = hipMallocPitch((void **)&dataCuda->filter, &pitch, sizeof(int), k);
	cudaTest(cudaStatus, "malloc filter");

	cudaStatus = hipMallocPitch((void **)&dataCuda->cubes1, &pitch, RECT_SIZE_3D*FRAME_CNT, k);
	cudaTest(cudaStatus, "malloc filter");
	cudaStatus = hipMallocPitch((void **)&dataCuda->cubes2, &pitch, RECT_SIZE_3D*FRAME_CNT, k);
	cudaTest(cudaStatus, "malloc filter");

	

	for (; i < frame->frame_count - FRAME_SKIP; i += FRAME_SKIP, j++) {
		shiftData(ref_data, frame->size);
		for (int k = 0; k < files_count; k++) {
			shiftData(data + indexFs(0, 0, frame->size, k), frame->size);
		}
		for (int k = FRAME_CNT / 2 + 1; k < FRAME_CNT; k++) {
			for (int l = 0; l < files_count; l++) {
				readFromFile(dataTmp, frame->size, streams[l]);
				readFromFile(dataTrash, frame->size / 2, streams[l]);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
				memcpy(data + indexFs(k, 0, frame->size, l), dataTmp, frame->size);
				//hipMemcpy(data + indexFs(k, 0, frame->size,l), dataTmp, frame->size, hipMemcpyHostToDevice);
			}
			readFromFile(dataTmp, frame->size, ref);
			readFromFile(dataTrash, frame->size / 2, ref);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			memcpy(ref_data + indexF(k, 0, frame->size), dataTmp, frame->size); //FIXME optimalize
																				//hipMemcpy(ref_data + indexF(k, 0, frame->size), dataTmp, frame->size, hipMemcpyHostToDevice);
		}
		double resSSIM, res3D;
		for (int l = 0; l < files_count; l++) {
			//cout<<"diff!!: "<<indexFs(0, 0, frame->size, l)<<endl;
			res3D = countSTVSSIM_CUDA(ref_data, data + indexFs(0, 0, frame->size, l), frame->size, frame->width,dataCuda);
			//resSSIM = countSSIM(ref_data[FRAME_CNT / 2], data[l][FRAME_CNT / 2], frame->size, frame->width);
			results[l][j] = res3D;
			cout << "3D: " << res3D << endl;
//<< " SSIM: " << resSSIM << " Total: " << results[l][j] 
			//cout << j << ": " << results[l][j] << endl;
		}
		//cout << results[j] << endl;

	}
	for (int i = 0; i < files_count; i++) {
		frames[i] = j;
	}
	return results;
}

double countSTVSSIM_CUDA(unsigned char * datain1, unsigned char * datain2, int size, int width,data_CUDA * dataCuda) {

	unsigned char * out = new unsigned char[RECT_SIZE];
	int T = ROOD_SIZE;


	unsigned char * filters = new unsigned char[RECT_SIZE_3D*FRAME_CNT * 4];// = new unsigned char ****[CHUNK_SIZE]; //generateFilters();

	unsigned char **** cube1 = new unsigned char ***[CHUNK_SIZE]; //generateCube();
	unsigned char **** cube2 = new unsigned char ***[CHUNK_SIZE]; //generateCube();

	int k = 0;
	/*for (int i = 0; i < CHUNK_SIZE; i++) {
	filters[i] = generateFilters();
	cube1[i] = generateCube();
	cube2[i] = generateCube();
	out[i] = new unsigned char[RECT_SIZE];
	}*/

	int rectCount = (size / width - RECT_SQRT_3D)*(width - RECT_SQRT_3D) / SKIP_SIZE / SKIP_SIZE / THREADS*THREADS;
	int blocks = rectCount / THREADS;
	if (rectCount < (size / width - RECT_SQRT_3D)*(width - RECT_SQRT_3D) / SKIP_SIZE / SKIP_SIZE) {
		blocks = rectCount / THREADS + 1;
		rectCount = (size / width - RECT_SQRT_3D)*(width - RECT_SQRT_3D) / SKIP_SIZE / SKIP_SIZE;
	}
	vector vct;
	int * filter = new int[rectCount];
	for (int i = 0; i < size / width - RECT_SQRT_3D; i += SKIP_SIZE) {
		for (int j = 0; j < width - RECT_SQRT_3D; j += SKIP_SIZE) {
			k = (i / SKIP_SIZE)*((width - RECT_SQRT_3D) / SKIP_SIZE) + j / SKIP_SIZE;
			
			getRect(datain1 + indexF(FRAME_CNT / 2, 0, size), i*width+j, width, out); //FIXME - was i??
																						//if (abs(vct.x) > abs(vct.y)) T = abs(vct.x); FIXME
																						//if (abs(vct.x) < abs(vct.y)) T = abs(vct.y);
			vct = countARPS(out, datain1 + indexF(FRAME_CNT / 2 - 1, 0, size), j, i, width, size / width, T);

			if ((vct.x > vct.y * 2 && vct.x*-1 < 2 * vct.y) || (vct.x < vct.y * 2 && vct.x*-1 > 2 * vct.y)) { //y=0
				filter[k] = 0;
			}
			else if ((vct.y > vct.x * 2 && vct.y > -2 * vct.x) || (vct.y < vct.x * 2 && vct.y < -2 * vct.x)) { //x=0
				filter[k] = 2;
			}
			else if ((vct.y > vct.x / 2 && vct.y < 2 * vct.x) || (vct.y < vct.x / 2 && vct.y > 2 * vct.x)) { //y=x
				filter[k] = 3;
			}
			else if ((vct.y > vct.x / -2 && vct.y < -2 * vct.x) || (vct.y <vct.x / -2 && vct.y>-2 * vct.x)) { //y=-x
				filter[k] = 1;
			}
			else if (vct.x == 0 && vct.y == 0) {
				filter[k] = 8;
			}
			else if (vct.x == vct.y * 2) { //exactly between 2 axes
				filter[k] = 4;
			}
			else if (vct.x == vct.y * -2) {
				filter[k] = 5;
			}
			else if (-2 * vct.x == vct.y) {
				filter[k] = 6;
			}
			else if (2 * vct.x == vct.y) {
				filter[k] = 7;
			}
			else {
				cout << "WUT - nonsense vector " << vct.x << " " << vct.y << endl;
			}

		}
	}

	/*for (int i = 0; i < size / width - RECT_SQRT_3D; i += SKIP_SIZE) {
	for (int j = 0; j < width - RECT_SQRT_3D; j += SKIP_SIZE) {
	generateCube_CUDA()
	fillCube(datain1, i*width + j, cubeTmp, width);
	}
	}*/

	hipError_t cudaStatus;
	size_t pitch;
	/*unsigned char * filters_CUDA=NULL;
	int * filter_CUDA=NULL;
	unsigned char * datain1_CUDA=NULL;
	unsigned char * datain2_CUDA=NULL;
	unsigned char * cubes1_CUDA=NULL;
	unsigned char * cubes2_CUDA=NULL;*/
	
	generateFilters(filters);

	cudaStatus = hipMemcpy((void*)dataCuda->filters, (const void*)filters, FRAME_CNT*RECT_SIZE_3D*4, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy filters");
	cudaStatus = hipMemcpy((void*)dataCuda->datain1, (void*)datain1, size*FRAME_CNT, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy data1");
	cudaStatus = hipMemcpy((void*)dataCuda->datain2, (void*)datain2, size*FRAME_CNT, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy data2");
	cudaStatus = hipMemcpy((void*)dataCuda->filter, (void*)filter, rectCount * sizeof(int), hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy filter");

	
	//dont know how to transfer struct to device
	SSIM3DKernel << <blocks, THREADS >> > (dataCuda->datain1, dataCuda->datain2, dataCuda->cubes1,dataCuda->cubes2,dataCuda->filters,dataCuda->tmpRes,dataCuda->filter,width, size / width);

	hipDeviceSynchronize();
	double * tmpRes2 = new double[rectCount];
	hipMemcpy(tmpRes2, dataCuda->tmpRes, rectCount*sizeof(double), hipMemcpyDeviceToHost);
	
	
	double res = countRes(tmpRes2, rectCount);
	delete[] tmpRes2;

	/*
	for (int l = 0; l < CHUNK_SIZE; l++) {
	for (int i = 0; i < FRAME_CNT; i++) {
	for (int j = 0; j < RECT_SQRT_3D; j++) {

	delete[] cube1[l][i][j];
	delete[] cube2[l][i][j];
	//delete[] filters[l][0][j];
	}
	delete[] cube1[l][i];
	delete[] cube2[l][i];
	//delete[] filters[l][i];
	}
	delete[] cube1[l];
	delete[] cube2[l];
	//delete[] filters[l];
	delete[] out[l];
	}
	delete[] cube1;
	delete[] cube2;
	//delete[] filters;
	delete[] out;

	for (int l = 0; l < CHUNK_SIZE; l++) {
	for (int i = 0; i < 4; i++) {
	for (int j = 0; j < FRAME_CNT; j++) {
	for (int k = 0; k < RECT_SQRT_3D; k++) {
	delete[] filters[l][i][j][k];
	}
	delete[] filters[l][i][j];
	}
	delete[] filters[l][i];
	}
	delete[] filters[l];
	}
	delete[] filters;
	*/

	return res;

}
__global__ void SSIM3DKernel(unsigned char * datain1, unsigned char * datain2, unsigned char * cubes1,unsigned char * cubes2,unsigned char * filters,double * tmpRes,unsigned char * filter, int width, int height) {
	int i = threadIdx.x;
	int j = blockIdx.x;
	int pos = j*THREADS + i;
	//data_CUDA* dataCuda=*dataCuda1;
	if (pos>=((height - RECT_SQRT) / SKIP_SIZE*(width - RECT_SQRT) / SKIP_SIZE)){
		return;
	}
	unsigned char * cube1=cubes1+RECT_SIZE_3D*FRAME_CNT*pos;
	//generateCube_CUDA(&cube1);
	unsigned char *cube2=cubes2+RECT_SIZE_3D*FRAME_CNT*pos;
	//generateCube_CUDA(&cube2);
	//cube1[100]=0;

//	datain1_CUDA[100] = 62;
	fillCube(datain1, j*SKIP_SIZE*width + i*SKIP_SIZE, cube1, width, height);
	fillCube(datain2, j*SKIP_SIZE*width + i*SKIP_SIZE, cube2, width, height);

	double res0 = countSSIM3D(filters, cube1, cube2);
	double res1 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 1, cube1, cube2);
	double res2 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 2, cube1, cube2);
	double res3 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 3, cube1, cube2);
	switch (filter[pos]) {
	case 0:
		tmpRes[pos] = res0;
		break;
	case 1:
		tmpRes[pos] = res1;
		break;
	case 2:
		tmpRes[pos] = res2;
		break;
	case 3:
		tmpRes[pos] = res3;
		break;
	case 4:
		tmpRes[pos] = (res0 + res3) / 2;
		break;
	case 5:
		tmpRes[pos] = (res0 + res1) / 2;
		break;
	case 6:
		tmpRes[pos] = (res1 + res2) / 2;
		break;
	case 7:
		tmpRes[pos] = (res2 + res3) / 2;
		break;
	case 8:
		tmpRes[pos] = (res0 + res1 + res2 + res3) / 4;
		break;
	}


}

__device__ double countSSIM3D(unsigned char * filter, unsigned char *  cube1, unsigned char *  cube2) {
	double muX = countMu(filter, cube1);
	double muY = countMu(filter, cube2);

	double deltaSqrX = countDeltaSqr(filter, cube1, muX);
	double deltaSqrY = countDeltaSqr(filter, cube2, muY);

	double delta = countDelta(filter, cube1, cube2, muX, muY);

	double ssim3D = ((2 * muX*muY + C1)*(2 * delta + C2)) / ((muX*muX + muY*muY + C1)*(deltaSqrX + deltaSqrY + C2));
	return ssim3D;
}
__device__ double countMu(unsigned char* filter, unsigned char* cube) {
	double res = 0,res2=0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res2 += filter[index(gamma, alpha, beta)];
				res += /*filter[index(gamma, alpha, beta)] * */cube[index(gamma, alpha, beta)];
				
			}
		}
		//printf("res: %d\n",res);
	}
	return res; /// (RECT_SQRT_3D*FRAME_CNT);
}

__device__ double countDeltaSqr(unsigned char* filter, unsigned char* cube, double mu) {
	double res = 0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res += filter[index(gamma, alpha, beta)] * (cube[index(gamma, alpha, beta)] - mu)*(cube[index(gamma, alpha, beta)] - mu);
			}
		}
	}
	return res / (RECT_SQRT_3D*FRAME_CNT);
}


__device__ double countDelta(unsigned char* filter, unsigned char* cube1, unsigned char* cube2, double muX, double muY) {
	double res = 0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res += filter[index(gamma, alpha, beta)] * (cube1[index(gamma, alpha, beta)] - muX)*(cube2[index(gamma, alpha, beta)] - muX);
			}
		}
	}
	return res / (RECT_SQRT_3D*FRAME_CNT);
}

//Generates 3D array used for SSIM 3D
__device__ void generateCube_CUDA(unsigned char** cube) {
	hipError_t cudaStatus;
	//	unsigned char* cube;
	size_t pitch;
	*cube =(unsigned char*)malloc(FRAME_CNT*RECT_SIZE_3D);
//	*cube = new unsigned char[FRAME_CNT*RECT_SIZE_3D];
	/*cudaStatus = hipMallocPitch((void **)&cube, &pitch, FRAME_CNT, RECT_SIZE_3D);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed 3!");
	return NULL;
	}*/

	//return cube;
}

//Fill 3D array with data of surroundings pixels
__device__ void fillCube(unsigned char * datain, int pos, unsigned char *& out, int width, int height) {
	
		for (int j = 0; j < RECT_SQRT_3D; j++) {
			for (int k = 0; k < RECT_SQRT_3D; k++) {
				for (int i = 0; i < FRAME_CNT; i++) {
					out[index(i, j, k)]=datain[indexF(i, pos + width*j, width*height)];
				// = datain[0];
				//out[0] = datain[0];
			}
		}
	}
}

//generate cube filters, vertical, horizontal and 2 inclined are being created
//receives already malloced referenco to an CUDA array
__host__ unsigned char * generateFilters(unsigned char*& out) {

	for (int i = 0; i < FRAME_CNT; i++) { //FIXME modify to create 2D array and memcpy
		for (int j = 0; j < RECT_SQRT_3D; j++) {
			for (int k = 0; k < RECT_SQRT_3D; k++) {
				if (j == RECT_SQRT_3D / 2) { //horizontal filter
					out[index(0, i, j, k)] = 1;
				}
				else {
					out[index(0, i, j, k)] = 0;
				}
				if (k == RECT_SQRT_3D / 2) { //vertical filter
					out[index(1, i, j, k)] = 1;
				}
				else {
					out[index(1, i, j, k)] = 0;
				}


				if (j == k) { //x=y filter
					out[index(2, i, j, k)] = 1;
				}
				else {
					out[index(2, i, j, k)] = 0;
				}
				if (k + j == RECT_SQRT_3D) { //x=-y filter
					out[index(3, i, j, k)] = 1;
				}
				else {
					out[index(3, i, j, k)] = 0;
				}
			}
		}
	}
	return out;
}



/*
vector countARPS(unsigned char * block, unsigned char * framePrev, int x, int y, int width, int height, int T) {
unsigned char * out = new unsigned char[RECT_SIZE];
getRect(framePrev, x*y, width, out);
int sad = countSAD(block, out);
vector vOut;
if (sad < ZERO_MVMT) {
vOut.x = 0;
vOut.y = 0;
return vOut;
}
map<int, int > past;
int xOrig = x;
int yOrig = y;
int res[5];
while (1) {
getRect(framePrev, x + y*width, width, out);
res[0] = countSAD(block, out);
if (x - T - RECT_SQRT / 2 > 0) {
getRect(framePrev, (x - T) + y*width, width, out);
res[3] = countSAD(block, out);
}
else res[3] = INT_MAX;
if (x + T + RECT_SQRT / 2< width) {
getRect(framePrev, (x + T) + y*width, width, out);
res[1] = countSAD(block, out);
}
else res[1] = INT_MAX;
if (y + T + RECT_SQRT / 2< height) {
getRect(framePrev, x + (y + T)*width, width, out);
res[2] = countSAD(block, out);
}
else res[2] = INT_MAX;
if (y - T - RECT_SQRT / 2 > 0) {
getRect(framePrev, x + (y - T)*width, width, out);
res[4] = countSAD(block, out);
}
else res[4] = INT_MAX;
int min = INT_MAX;
int minPos = 0;
for (int i = 0; i < 5; i++) {
if (res[i] < min) {
min = res[i];
minPos = i;
}
}
if (minPos == 0) {
vOut.x = x - xOrig;
vOut.y = y - yOrig;
delete[] out;
return vOut;
}
switch (minPos) {
case 1:
x += T;
break;
case 2:
y += T;
break;
case 3:
x -= T;
break;
case 4:
y -= T;
break;
}
}
}
*/
__device__ __host__ void shiftData(unsigned char * data, int size) {
	for (int i = 0; i < FRAME_CNT / 2 + 1; i++) {
#if defined(__CUDA_ARCH__)
		hipMemcpy(data + indexF(i, 0, size), data + indexF(i + FRAME_CNT / 2, 0, size), size, hipMemcpyDeviceToDevice);
#else
		memcpy(data + indexF(i, 0, size), data + indexF(i + FRAME_CNT / 2, 0, size), size);
#endif
	}

}

__device__ __host__ int countSAD(unsigned char * rect1, unsigned  char * rect2) {
	int sad = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		sad += abs(rect1[i] - rect2[i]);
	}
	return sad;
}

//used for flattening of arrays because CUDA cannot work with multi dimensional arrays, FRAME_CNT frames
__device__  __host__  inline int indexF(const int x, const int y, const int size) {
	//cout<<"F : "<<x * size + y<<endl;
	return x * size + y;
}

//used for flattening of arrays because CUDA cannot work with multi dimensional arrays, FRAME_CNT frames with files_count files
__device__  __host__  inline int indexFs(const int x, const int y, const int size, const int file_index) {
	//cout<<"FS: "<<file_index * size * FRAME_CNT + x * size  + y<<endl;
	return file_index * size * FRAME_CNT + x * size  + y;
}

//used for flattening of arrays because CUDA cannot work with multi dimensional arrays, cube
__device__  __host__ inline int index(const int x, const int y, const int z) {
	return y * RECT_SQRT_3D*FRAME_CNT + z * FRAME_CNT + x;
}

//used for flattening of arrays because CUDA cannot work with multi dimensional arrays, filter
__device__  __host__ inline int index(const int x, const int y, const int z, const int aa) {
	return x * RECT_SIZE_3D*FRAME_CNT + y * RECT_SIZE_3D + z *RECT_SQRT_3D + aa;
}
