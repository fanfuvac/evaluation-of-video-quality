#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "stvssim.cuh"
#include "stvssim.h"
#include "ssim.cuh"
#include "SSIM.h"
#include <stdlib.h>     /* abs */
#include <map>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;


void cudaTest(hipError_t cudaStatus, string descr) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%d", int(cudaStatus));
		fprintf(stderr, strcat("cuda failed: ", descr.c_str()));
		exit(-1);
	}

}

///count STVSSIM metric for all files 
double ** countMetricSTVSSIM_CUDA(FILE ** streams, FILE * ref, int files_count, PictureData * frame, double ** results, int *& frames) {
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(-1);
		return NULL;
	}
	int rec;
	size_t pitch;

	unsigned char * ref_data = new unsigned char[FRAME_CNT*frame->size];
	unsigned char * data = new unsigned char[FRAME_CNT*frame->size*files_count];
	unsigned char * dataTrash = new unsigned char[frame->size / 2];
	unsigned char * dataTmp = new unsigned char[frame->size];

	for (int i = FRAME_CNT / 2; i < FRAME_CNT; i++) {
		for (int j = 0; j < files_count; j++) {
			readFromFile(dataTmp, frame->size, streams[j]);
			readFromFile(dataTrash, frame->size / 2, streams[j]);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			memcpy(data + indexFs(i, 0, frame->size, j), dataTmp, frame->size);
		}
		readFromFile(dataTmp, frame->size, ref);
		readFromFile(dataTrash, frame->size / 2, ref);
		memcpy(ref_data + indexF(i, 0, frame->size), dataTmp, frame->size);
	}

	int i = FRAME_SKIP;
	int j = 0;
	data_CUDA * dataCuda = new data_CUDA;

	cudaStatus = hipMallocPitch((void **)&dataCuda->filters, &pitch, RECT_SIZE_3D, FRAME_CNT * 4);//4 different filters with size of one cube
	cudaTest(cudaStatus, "malloc filters");
	cudaStatus = hipMallocPitch((void **)&dataCuda->datain1, &pitch, FRAME_CNT, frame->size);
	cudaTest(cudaStatus, "malloc datain1");
	cudaStatus = hipMallocPitch((void **)&dataCuda->datain2, &pitch, FRAME_CNT, frame->size);
	int k = (frame->size / frame->width - RECT_SQRT) / SKIP_SIZE*(frame->width - RECT_SQRT) / SKIP_SIZE;
	cudaStatus = hipMallocPitch((void **)&dataCuda->tmpRes, &pitch, sizeof(double), k);
	cudaTest(cudaStatus, "malloc datain2");
	cudaStatus = hipMallocPitch((void **)&dataCuda->filter, &pitch, sizeof(int), k);
	cudaTest(cudaStatus, "malloc filter");

	cudaStatus = hipMallocPitch((void **)&dataCuda->cubes1, &pitch, RECT_SIZE_3D*FRAME_CNT, k);
	cudaTest(cudaStatus, "malloc filter");
	cudaStatus = hipMallocPitch((void **)&dataCuda->cubes2, &pitch, RECT_SIZE_3D*FRAME_CNT, k);
	cudaTest(cudaStatus, "malloc filter");

	//preparation for SSIM GPU part
	unsigned char * dataRef;
	cudaStatus = hipMalloc((void **)&dataRef, frame->width*frame->height);
	cudaTest(cudaStatus, "malloc dataRef");

	unsigned char ** dataGPU = new unsigned char *[files_count];
	//allocated the device memory for source array  
	for (int i = 0; i < files_count; i++) {
		cudaStatus = hipMalloc((void **)&dataGPU[i], frame->width*frame->height);
		cudaTest(cudaStatus, "malloc data GPU SSIM");
	}

	unsigned char ** rects = new unsigned char *[files_count];
	for (int i = 0; i < 2; i++) { //if replaced with files_count it can start kernels in parallel
		cudaStatus = hipMallocPitch((void **)&rects[i], &pitch, RECT_SIZE, frame->width*frame->height / SKIP_SIZE / SKIP_SIZE);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed 3!");
			return NULL;
		}
	}

	double * resultsFrame;
	cudaStatus = hipMallocPitch((void**)&resultsFrame, &pitch, sizeof(double), frame->width*frame->height / SKIP_SIZE / SKIP_SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 5!");

		return NULL;
	}
	//end of preparation for GPU SSIM

	for (; i < frame->frame_count - FRAME_SKIP-1; i += FRAME_SKIP, j++) {
		shiftData(ref_data, frame->size);
		for (int k = 0; k < files_count; k++) {
			shiftData(data + indexFs(0, 0, frame->size, k), frame->size);
		}
		for (int k = FRAME_CNT / 2 + 1; k < FRAME_CNT; k++) {
			for (int l = 0; l < files_count; l++) {
				readFromFile(dataTmp, frame->size, streams[l]);
				readFromFile(dataTrash, frame->size / 2, streams[l]);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
				memcpy(data + indexFs(k, 0, frame->size, l), dataTmp, frame->size);
				//hipMemcpy(data + indexFs(k, 0, frame->size,l), dataTmp, frame->size, hipMemcpyHostToDevice);
			}
			readFromFile(dataTmp, frame->size, ref);
			readFromFile(dataTrash, frame->size / 2, ref);//when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			memcpy(ref_data + indexF(k, 0, frame->size), dataTmp, frame->size); //FIXME optimalize
																				//hipMemcpy(ref_data + indexF(k, 0, frame->size), dataTmp, frame->size, hipMemcpyHostToDevice);
		}
		double resSSIM=0, res3D;
		cout << "file" << "\t" << "3D" << "\t" << "SSIM" << "\t" << "total" << endl;
		for (int l = 0; l < files_count; l++) {
			res3D = countSTVSSIM_CUDA(ref_data, data + indexFs(0, 0, frame->size, l), frame->size, frame->width, dataCuda);
			
			//now we can either use CPU or GPU version of SSIM algorithm
			//resSSIM = countSSIM(ref_data+indexF(FRAME_CNT / 2,0,frame->size), data+ indexFs(FRAME_CNT / 2, 0, frame->size, l), frame->size, frame->width);
			resSSIM=countSSIM(ref_data + indexF(FRAME_CNT / 2, 0, frame->size), data + indexFs(FRAME_CNT / 2, 0, frame->size, l), dataRef, dataGPU[l], rects[0], rects[1], frame->size, frame->width, resultsFrame);
			results[l][j] = res3D*resSSIM;
			cout << l << "\t" << res3D << "\t" << resSSIM << "\t" << results[l][j] << endl;
			//cout << "3D: " << res3D << " SSIM: " << resSSIM << " Total: " << results[l][j] << endl;
		}
	}
	for (int i = 0; i < files_count; i++) {
		frames[i] = j;
	}
	return results;
}

/*!counting STVSSIM for given set of frames
 @param[in] datain1,datain2 input datasets
 @param[in] size size in bytes of one image
 @param[in] width width of one image
 @param[in] dataCuda struct with preallocated space on CUDA device
*/
double countSTVSSIM_CUDA(unsigned char * datain1, unsigned char * datain2, int size, int width, data_CUDA * dataCuda) {

	unsigned char * out = new unsigned char[RECT_SIZE];
	int T = ROOD_SIZE;


	unsigned char * filters = new unsigned char[RECT_SIZE_3D*FRAME_CNT * 4];// = new unsigned char ****[CHUNK_SIZE]; //generateFilters();

	//unsigned char **** cube1 = new unsigned char ***[CHUNK_SIZE]; //generateCube();
	//unsigned char **** cube2 = new unsigned char ***[CHUNK_SIZE]; //generateCube();

	int k = 0;

	int rectCount = (int)((size / width - RECT_SQRT_3D) / SKIP_SIZE + 1)*	(int)((width - RECT_SQRT_3D) / SKIP_SIZE + 1) / THREADS*THREADS;
	int blocks = rectCount / THREADS;
	if (rectCount < (int)((size / width - RECT_SQRT_3D) / SKIP_SIZE + 1)*	(int)((width - RECT_SQRT_3D) / SKIP_SIZE + 1)) {
		blocks = rectCount / THREADS + 1;
		rectCount = (int)((size / width - RECT_SQRT_3D) / SKIP_SIZE + 1)*	(int)((width - RECT_SQRT_3D) / SKIP_SIZE + 1);
	}
	vector vct;
	int * filter = new int[rectCount];
	for (int i = 0; i < size / width - RECT_SQRT_3D; i += SKIP_SIZE) {
		for (int j = 0; j < width - RECT_SQRT_3D; j += SKIP_SIZE, k++) {
			getRect(datain1 + indexF(FRAME_CNT / 2, 0, size), i*width + j, width, out); 
			
			vct = countARPS(out, datain1 + indexF(FRAME_CNT / 2 - 1, 0, size), j, i, width, size / width, T);
			//if (abs(vct.x) > abs(vct.y)) T = abs(vct.x); FIXME
			//if (abs(vct.x) < abs(vct.y)) T = abs(vct.y);
			
			if ((vct.x > vct.y * 2 && vct.x*-1 < 2 * vct.y) || (vct.x < vct.y * 2 && vct.x*-1 > 2 * vct.y)) { //y=0
				filter[k] = 0;
			}
			else if ((vct.y > vct.x * 2 && vct.y > -2 * vct.x) || (vct.y < vct.x * 2 && vct.y < -2 * vct.x)) { //x=0
				filter[k] = 2;
			}
			else if ((vct.y > vct.x / 2 && vct.y < 2 * vct.x) || (vct.y < vct.x / 2 && vct.y > 2 * vct.x)) { //y=x
				filter[k] = 3;
			}
			else if ((vct.y > vct.x / -2 && vct.y < -2 * vct.x) || (vct.y <vct.x / -2 && vct.y>-2 * vct.x)) { //y=-x
				filter[k] = 1;
			}
			else if (vct.x == 0 && vct.y == 0) {
				filter[k] = 8;
			}
			else if (vct.x == vct.y * 2) { //exactly between 2 axes
				filter[k] = 4;
			}
			else if (vct.x == vct.y * -2) {
				filter[k] = 5;
			}
			else if (-2 * vct.x == vct.y) {
				filter[k] = 6;
			}
			else if (2 * vct.x == vct.y) {
				filter[k] = 7;
			}
			else {
				cout << "WUT - nonsense vector " << vct.x << " " << vct.y << endl;
			}
			//printf("vct: %d %d %d %d\n",vct.x,vct.y,k,filter[k]);
		}
	}

	hipError_t cudaStatus;
	size_t pitch;
	
	generateFilters(filters);

	cudaStatus = hipMemcpy((void*)dataCuda->filters, (const void*)filters, FRAME_CNT*RECT_SIZE_3D * 4, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy filters");
	cudaStatus = hipMemcpy((void*)dataCuda->datain1, (void*)datain1, size*FRAME_CNT, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy data1");
	cudaStatus = hipMemcpy((void*)dataCuda->datain2, (void*)datain2, size*FRAME_CNT, hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy data2");
	cudaStatus = hipMemcpy((void*)dataCuda->filter, (void*)filter, rectCount * sizeof(int), hipMemcpyHostToDevice);
	cudaTest(cudaStatus, "memcpy filter");


	//dont know how to transfer struct to device
	SSIM3DKernel << <blocks, THREADS >> > (dataCuda->datain1, dataCuda->datain2, dataCuda->cubes1, dataCuda->cubes2, dataCuda->filters, dataCuda->tmpRes, dataCuda->filter, width, size / width);

	hipDeviceSynchronize();
	double * tmpRes2 = new double[rectCount];
	hipMemcpy(tmpRes2, dataCuda->tmpRes, rectCount * sizeof(double), hipMemcpyDeviceToHost);


	double res = countRes(tmpRes2, rectCount);
	delete[] tmpRes2;
	delete[] filters;

	return res;

}

/*!Kernel which is being executed on CUDA device, sounting SSIM-3D part
 @param[in] datain1,datain2 input datasets, allocated on GPU card
 @param cubes1,cubes2 allocated space for data, orriginaly as 3D array, here only 1D array due to CUDA
 @param filters similar to cubes1, used for filtering the data
 @param tmpRes preallocated space on CUDA device used for temporary stored results
 @param filter list of filters being used for each rectangle
 @param width,height size of respective dimension of the frame
*/
__global__ void SSIM3DKernel(unsigned char * datain1, unsigned char * datain2, unsigned char * cubes1, unsigned char * cubes2, unsigned char * filters, double * tmpRes, int * filter, int width, int height) {
	int i = threadIdx.x;
	int j = blockIdx.x;
	int pos = j*blockDim.x + i;
	if (pos >= ((int)((height - RECT_SQRT_3D) / SKIP_SIZE + 1)*	(int)((width - RECT_SQRT_3D) / SKIP_SIZE + 1))) {
		return;
	}
	unsigned char * cube1 = cubes1 + RECT_SIZE_3D*FRAME_CNT*pos;
	unsigned char *cube2 = cubes2 + RECT_SIZE_3D*FRAME_CNT*pos;
	int leftover = (width - RECT_SQRT_3D) % SKIP_SIZE;
	int line = width - RECT_SQRT_3D - leftover+ SKIP_SIZE; //effective size of the line
	int start = (pos*SKIP_SIZE) % ((line)) + (pos*SKIP_SIZE) / line * width*SKIP_SIZE;
	//printf("%d %d: %d ",blockIdx.x, threadIdx.x, start);
	fillCube(datain1, start, cube1, width, height);
	fillCube(datain2, start, cube2, width, height);

	double res0 = countSSIM3D(filters, cube1, cube2);
	double res1 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 1, cube1, cube2);
	double res2 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 2, cube1, cube2);
	double res3 = countSSIM3D(filters + FRAME_CNT*RECT_SIZE_3D * 3, cube1, cube2);
	//printf("%d %d %d: %f %f %f %f %d\n", start, blockIdx.x, threadIdx.x, res0, res1, res2, res3, filter[pos]);
	switch (filter[pos]) {
	case 0:
		tmpRes[pos] = res0;
		break;
	case 1:
		tmpRes[pos] = res1;
		break;
	case 2:
		tmpRes[pos] = res2;
		break;
	case 3:
		tmpRes[pos] = res3;
		break;
	case 4:
		tmpRes[pos] = (res0 + res3) / 2;
		break;
	case 5:
		tmpRes[pos] = (res0 + res1) / 2;
		break;
	case 6:
		tmpRes[pos] = (res1 + res2) / 2;
		break;
	case 7:
		tmpRes[pos] = (res2 + res3) / 2;
		break;
	case 8:
		tmpRes[pos] = (res0 + res1 + res2 + res3) / 4;
		break;
	}


}


/*!Computation of SSIM-3D part, computes one cube, being executed on device
 @param[in] filter respective filter which will be used for this function
 @param cubes1,cubes2 data with cumputed part of framesorriginaly as 3D array, here only 1D array due to CUDA
*/
__device__ double countSSIM3D(unsigned char * filter, unsigned char *  cube1, unsigned char *  cube2) {
	double muX = countMu(filter, cube1);
	//printf("%d %d: %.0f ",blockIdx.x, threadIdx.x, muX);
	double muY = countMu(filter, cube2);

	double deltaSqrX = countDeltaSqr(filter, cube1, muX);
	double deltaSqrY = countDeltaSqr(filter, cube2, muY);

	double delta = countDelta(filter, cube1, cube2, muX, muY);

	double ssim3D = ((2 * muX*muY + C1)*(2 * delta + C2)) / ((muX*muX + muY*muY + C1)*(deltaSqrX + deltaSqrY + C2));
	return ssim3D;
}
__device__ double countMu(unsigned char* filter, unsigned char* cube) {
	double res = 0, res2 = 0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res2 += filter[index(gamma, alpha, beta)];
				res += filter[index(gamma, alpha, beta)] * cube[index(gamma, alpha, beta)];
				//printf("%.0f ",res);

			}
		}
		//printf("res: %d\n",res);
	}
	return res / (RECT_SQRT_3D*FRAME_CNT);
}

__device__ double countDeltaSqr(unsigned char* filter, unsigned char* cube, double mu) {
	double res = 0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res += filter[index(gamma, alpha, beta)] * (cube[index(gamma, alpha, beta)] - mu)*(cube[index(gamma, alpha, beta)] - mu);
			}
		}
	}
	return res / (RECT_SQRT_3D*FRAME_CNT);
}


__device__ double countDelta(unsigned char* filter, unsigned char* cube1, unsigned char* cube2, double muX, double muY) {
	double res = 0;
	for (int alpha = 0; alpha < RECT_SQRT_3D; alpha++) {
		for (int beta = 0; beta < RECT_SQRT_3D; beta++) {
			for (int gamma = 0; gamma < FRAME_CNT; gamma++) {
				res += filter[index(gamma, alpha, beta)] * (cube1[index(gamma, alpha, beta)] - muX)*(cube2[index(gamma, alpha, beta)] - muX);
			}
		}
	}
	return res / (RECT_SQRT_3D*FRAME_CNT);
}

///Generates 3D array used for SSIM 3D
__device__ void generateCube_CUDA(unsigned char** cube) {
	hipError_t cudaStatus;
	//	unsigned char* cube;
	size_t pitch;
	*cube = (unsigned char*)malloc(FRAME_CNT*RECT_SIZE_3D);


	//return cube;
}

///Fill 3D array with data of surroundings pixels
__device__ void fillCube(unsigned char * datain, int pos, unsigned char *& out, int width, int height) {
	for (int i = 0; i < FRAME_CNT; i++) {
		for (int j = 0; j < RECT_SQRT_3D; j++) {
			for (int k = 0; k < RECT_SQRT_3D; k++) {
				out[index(i, j, k)] = datain[indexF(i, pos + width*j + k, width*height)];
			}
		}
	}
}
/*!generate cube filters, vertical, horizontal and 2 inclined are being created
 @param out already malloced referenco to an CUDA array
*/
__host__ unsigned char * generateFilters(unsigned char*& out) {

	for (int i = 0; i < FRAME_CNT; i++) { //FIXME modify to create 2D array and memcpy
		for (int j = 0; j < RECT_SQRT_3D; j++) {
			for (int k = 0; k < RECT_SQRT_3D; k++) {
				if (j == RECT_SQRT_3D / 2) { //horizontal filter
					out[index(0, i, j, k)] = 1;
				}
				else {
					out[index(0, i, j, k)] = 0;
				}
				if (k == RECT_SQRT_3D / 2) { //vertical filter
					out[index(1, i, j, k)] = 1;
				}
				else {
					out[index(1, i, j, k)] = 0;
				}


				if (j == k) { //x=y filter
					out[index(2, i, j, k)] = 1;
				}
				else {
					out[index(2, i, j, k)] = 0;
				}
				if (k + j == RECT_SQRT_3D) { //x=-y filter
					out[index(3, i, j, k)] = 1;
				}
				else {
					out[index(3, i, j, k)] = 0;
				}
			}
		}
	}
	return out;
}



/*
//implement this for CUDA? Probably would not be effective because of too many if and splitting of the code
vector countARPS(unsigned char * block, unsigned char * framePrev, int x, int y, int width, int height, int T) {
}
*/
__device__ __host__ void shiftData(unsigned char * data, int size) {
	for (int i = 0; i < FRAME_CNT / 2 + 1; i++) {
#if defined(__CUDA_ARCH__)
		hipMemcpy(data + indexF(i, 0, size), data + indexF(i + FRAME_CNT / 2, 0, size), size, hipMemcpyDeviceToDevice);
#else
		memcpy(data + indexF(i, 0, size), data + indexF(i + FRAME_CNT / 2, 0, size), size);
#endif
	}

}

__device__ __host__ int countSAD(unsigned char * rect1, unsigned  char * rect2) {
	int sad = 0;
	for (int i = 0; i < RECT_SIZE; i++) {
		sad += abs(rect1[i] - rect2[i]);
	}
	return sad;
}

///used for flattening of arrays because CUDA cannot work with multi dimensional arrays, FRAME_CNT frames
__device__  __host__  inline int indexF(const int x, const int y, const int size) {
	return x * size + y;
}

///used for flattening of arrays because CUDA cannot work with multi dimensional arrays, FRAME_CNT frames with files_count files
__device__  __host__  inline int indexFs(const int x, const int y, const int size, const int file_index) {
	return file_index * size * FRAME_CNT + x * size + y;
}

///used for flattening of arrays because CUDA cannot work with multi dimensional arrays, cube
__device__  __host__ inline int index(const int x, const int y, const int z) {
	return x * RECT_SIZE_3D + y * RECT_SQRT_3D + z;
}

///used for flattening of arrays because CUDA cannot work with multi dimensional arrays, filter
__device__  __host__ inline int index(const int x, const int y, const int z, const int aa) {
	return x * RECT_SIZE_3D*FRAME_CNT + y * RECT_SIZE_3D + z *RECT_SQRT_3D + aa;
}
