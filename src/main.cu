#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "SSIM.h"   
#include "main.h"   
#include "psnr.h"   
#include "stvssim.h"
#include "ssim.cuh"
#include <omp.h>

using namespace std;
int compare(const void * a, const void * b)
{
	return (*(double*)a - *(double*)b);
}
PictureData *getVideoInfo(string path) {
	PictureData * data = new PictureData;
	cout << path.c_str() << endl;
	string cmd = "ffprobe -v error -of flat=s=_  -select_streams v:0 -show_entries stream=width,height,r_frame_rate -show_entries format=duration,nb_frames -of default=noprint_wrappers=1:nokey=1 " + path;
	cout << cmd.c_str() << endl;
	//string cmd="ffprobe -v error -of flat=s=_ -select_streams v:0 -show_entries stream=width,height,nb_frames -of default=noprint_wrappers=1:nokey=1 "+path;
	string cmd2 = "ffprobe - select_streams v - show_streams" + path + " 2> NUL";

#ifdef __linux__
	FILE *stream = popen(cmd.c_str(), "r");
#else 
	FILE *stream = _popen(cmd.c_str(), "r");
#endif
	char buffer[50];
	fgets(buffer, 10, stream);
	data->width = atoi(buffer);
	fgets(buffer, 10, stream);
	data->height = atoi(buffer);
	fgets(buffer, 20, stream);
	string tmp = buffer;
	int pos = tmp.find('/');
	int fps1 = atoi(buffer);
	double fps2 = atoi(tmp.substr(pos + 1).c_str());
	double fps = fps1 / fps2;
	cout << fps << endl;
	fgets(buffer, 20, stream);
	//cout << buffer << endl;


	double len = atof(buffer);

	cout << len*fps << endl;
	data->frame_count = len*fps;
	data->size = data->width*data->height;
	//else data->frame_count = 3121;//181250; // 7100;//3121;//1359;//7192;
	return data;
}
FILE * startFFmpeg(string path) {
	FILE *stream;
#ifdef __linux__
	string cmd = "ffmpeg -i " + path + " -f image2pipe -pix_fmt yuv420p -vcodec rawvideo - 2>/dev/null";
	cout << cmd << endl;
	stream = popen(cmd.c_str(), "rb");
#else 
	string cmd = "ffmpeg -i " + path + " -f image2pipe -threads 3  -pix_fmt yuv420p -vcodec rawvideo - 2>NUL";
	//-c:v h264_qsv
	stream = _popen(cmd.c_str(), "rb");
#endif
	cout << cmd.c_str() << endl;


	return stream;
}

void shiftData(unsigned char ** data, int size) {
	for (int i = 0; i < FRAME_CNT / 2 + 1; i++) {
		memcpy(data[i], data[i + FRAME_CNT / 2], size);
	}

}

double ** countMetric(FILE ** streams, FILE * ref, int files_count, PictureData * frame, string type, double ** results) {

	//double ** results = new double*[files_count];
	unsigned char *** data = new unsigned char **[files_count];
	unsigned char ** dataRef = new unsigned char *[CHUNK_SIZE];
	unsigned char * dataTrash = new unsigned char[frame->width*frame->height / 2];
	int frames = frame->frame_count;
	int rec;
	for (int k = 0; k < files_count; k++) {
		data[k] = new unsigned char *[CHUNK_SIZE];
		//results[k] = new double [frame->frame_count];
		for (int j = 0; j < CHUNK_SIZE; j++) {
			data[k][j] = new unsigned char[frame->width*frame->height];

		}
	}
	for (int j = 0; j < CHUNK_SIZE; j++) {
		dataRef[j] = new unsigned char[frame->width*frame->height];
	}

	for (int i = 0; i < frame->frame_count / CHUNK_SIZE; i++) {
		for (int j = 0; j < CHUNK_SIZE; j++) {
			rec = fread(dataRef[j], 1, frame->width*frame->height, ref);
			if (rec != frame->width*frame->height) {
				cout << "error" << endl;
				return NULL;
			}
			rec = fread(dataTrash, 1, frame->width*frame->height / 2, ref); //when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			if (rec != frame->width*frame->height / 2) {
				cout << "error" << endl;
				return NULL;
			}
			for (int k = 0; k < files_count; k++) {
				int rec = fread(data[k][j], 1, frame->width*frame->height, streams[k]);
				if (rec != frame->width*frame->height) {
					cout << "error" << endl;
					return NULL;
				}
				rec = fread(dataTrash, 1, frame->width*frame->height / 2, streams[k]); //when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
				if (rec != frame->width*frame->height / 2) {
					cout << "error" << endl;
					return NULL;
				}
			}


		}
		omp_set_num_threads(CHUNK_SIZE);
		for (int k = 0; k < files_count; k++) {
#pragma omp parallel for 
			for (int j = 0; j < CHUNK_SIZE; j++) {

				if (string(type) == string("SSIM")) results[k][j + i*CHUNK_SIZE] = countSSIM(dataRef[j], data[k][j], frame->width*frame->height, frame->width);

				else results[k][j + i*CHUNK_SIZE] = countPSNR(dataRef[j], data[k][j], frame->width*frame->height);
				//cout << j+i * CHUNK_SIZE << " " << results[k][j+i*CHUNK_SIZE] << endl;
			}
		}
	}
	for (int j = 0; j < frame->frame_count % CHUNK_SIZE; j++) {
		rec = fread(dataRef[j], 1, frame->width*frame->height, ref);
		fread(dataTrash, 1, frame->width*frame->height/2, ref);
		
		//fseek(ref, frame->width*frame->height / 2, SEEK_CUR); //skip others except Y channel
		if (rec != frame->width*frame->height/2) {
			cout << "error" << endl;
			return NULL;
		}
	}
	for (int k = 0; k < files_count; k++) {
		for (int j = 0; j < frame->frame_count % CHUNK_SIZE; j++) {
			rec = fread(data[k][j], 1, frame->width*frame->height, streams[k]);
			fread(dataTrash, 1, frame->width*frame->height/2, streams[k]);
			
			//fseek(streams[k], frame->width*frame->height / 2, SEEK_CUR); //skip others except Y channel
			if (rec != frame->width*frame->height) {
				cout << "error" << endl;
				return NULL;
			}

			if (string(type) == string("SSIM"))  results[k][frame->frame_count - frame->frame_count % CHUNK_SIZE + j] = countSSIM(dataRef[j], data[k][j], frame->width*frame->height, frame->width);
			else results[k][frame->frame_count - frame->frame_count % CHUNK_SIZE + j] = countPSNR(dataRef[j], data[k][j], frame->width*frame->height);
			//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
		}
	}

	//delete frame->data;
	delete frame;
	return results;
}


int main(int argc, char ** argv) {

	string reference;
	string file1, file2;
	string type;
	int gpu = 0;
	string  * files = new string[MAX_FILES];
	int files_count = 0;
	if (argc < 6) { // Check the value of argc. If not enough parameters have been passed, inform user and exit.
		cout << argc << endl;
		cout << "Usage is -r <reference file> -in <first video to compare> -in <second video to compare> [-type]\n"; // Inform the user of how to use the program
																													 //std::cin.get();
		exit(0);
	}
	else { // if we got enough parameters...

		//std::cout << argv[0];
		for (int i = 1; i < argc; i++) { /* We will iterate over argv[] to get the parameters stored inside.
										 * Note that we're starting on 1 because we don't need to know the
										 * path of the program, which is stored in argv[0] */
			if (i + 1 != argc) { // Check that we haven't finished parsing already
				if (string(argv[i]) == string("-r")) {
					// We know the next argument *should* be the filename:
					reference = argv[i + 1];
					//std::cout << reference << endl;
				}
				else if (string(argv[i]) == string("-in")) {
					files[files_count] = string(argv[i + 1]);
					//cout << files[files_count].c_str() << endl;
					files_count++;
				}
				/*else if (string(argv[i]) == string("-in2")) {
				file2 = string(argv[i + 1]);
				}*/
				else if (string(argv[i]) == string("-type")) {
					type = string(argv[i + 1]);

				}
				
				else {
					//cout << "Not enough or invalid arguments, please try again.\n";
					//Sleep(2000);
					//exit(0);
				}
				//std::cout << argv[i] << " ";
			}
			else if (i != argc){
				if (string(argv[i]) == string("CUDA")) { //we will use CUDA computation
					gpu = 1;
				}
			}
		}
	}

	const int MAX_BUFFER = 2048000;
	PictureData * frame;
	FILE ** streams;
	FILE * ref;
	streams = new FILE *[files_count];
	double ** results = new double *[files_count];
	frame = getVideoInfo(reference);
	ref = startFFmpeg(reference);
	for (int i = 0; i < files_count; i++) {
		frame = getVideoInfo(files[i]);
		streams[i] = startFFmpeg(files[i]);
		results[i] = new double[frame->frame_count];
	}


	int rec;
	double *sum = new double[files_count];
	int * frames = new int[files_count];
	for (int i = 0; i < files_count; i++) {
		frames[i] = frame->frame_count;
		sum[i] = 0;
	}
	if (gpu == 1) {
		countCUDA(streams, ref, files_count, frame, type, results);
	}

	else if (string(type) == string("STVSSIM")) {

		countMetricSTVSSIM(streams, ref, files_count, frame, type, results);
		//results= new double *[files_count];
		unsigned char ** ref_data = new unsigned char *[FRAME_CNT];
		unsigned char *** data = new unsigned char **[FRAME_CNT];
		for (int k = 0; k < files_count; k++) {
			//results[k] = new double[frame->frame_count];
			data[k] = new unsigned char *[FRAME_CNT];
			for (int j = 0; j < FRAME_CNT; j++) {
				data[k][j] = new unsigned char[frame->size];
			}
		}
		for (int j = 0; j < FRAME_CNT; j++) {
			ref_data[j] = new unsigned char[frame->size];
		}
		unsigned char * tmp = new  unsigned char[frame->size * 3];

		for (int i = FRAME_CNT / 2; i < FRAME_CNT; i++) {
			for (int j = 0; j < files_count; j++) {
				rec = fread(tmp, 1, frame->width*frame->height * 3, streams[j]);
				if (rec != frame->width*frame->height * 3) {
					cout << "error" << endl;
					return -1;
				}
				//getLuma(tmp, data[j][i], frame->size);
			}

			rec = fread(tmp, 1, frame->width*frame->height * 3, ref);
			if (rec != frame->width*frame->height * 3) {
				cout << "error2" << endl;
				return -1;
			}
			//getLuma(tmp, ref_data[i], frame->size);
		}

		int i = FRAME_SKIP;
		int j = 0;

		for (; i < frame->frame_count - FRAME_SKIP; i += FRAME_SKIP, j++) {
			shiftData(ref_data, frame->size);
			for (int k = 0; k < files_count; k++) {
				shiftData(data[k], frame->size);
			}
			for (int k = 0; k < FRAME_SKIP; k++) {
				for (int l = 0; l < files_count; l++) {
					rec = fread(tmp, 1, frame->width*frame->height * 3, streams[l]);
					if (rec != frame->width*frame->height * 3) {
						cout << "error" << endl;
						return -1;
					}
					//getLuma(tmp, data[l][k], frame->size);
				}
			}
			for (int l = 0; l < files_count; l++) {
				results[l][j] = countSTVSSIM(ref_data, data[l], frame->width*frame->height, frame->width);
				cout << j << ": " << results[l][j] << endl;
			}
			//cout << results[j] << endl;

		}
		for (int i = 0; i < files_count; i++) {
			frames[i] = j;
		}
		
	}
	else {
		countMetric(streams, ref, files_count, frame, type, results); //SSIM, PSNR
		
	}

	for (int j = 0; j < files_count; j++) {
		cout << "input file number: " << j << endl;
		for (int i = 0; i < frame->frame_count; i++) {
			cout << i << " " << results[j][i] << endl;
			if (isfinite(results[j][i]))
				sum[j] += results[j][i];
			else frames[j]--;
		}

	}
	for (int i = 0; i < files_count; i++) {
		cout << "AVG: " << sum[i] / frames[i] << endl;
		qsort(results[i], frames[i], sizeof(double), compare);
		cout << "Median: " << results[i][frames[i] / 2] << endl;
	}
}