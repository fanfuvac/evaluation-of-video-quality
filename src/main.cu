#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <math.h>  
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "SSIM.h"   
#include "main.h"   
#include "psnr.h"   
#include "stvssim.h"
#include "stvssim.cuh"
#include "ssim.cuh"
#include <omp.h>

using namespace std;
int compare(const void * a, const void * b)
{
	return (*(double*)a - *(double*)b);
}
PictureData *getVideoInfo(string path) {
	PictureData * data = new PictureData;
	cout << path.c_str() << endl;
	string cmd = "ffprobe -v error -of flat=s=_  -select_streams v:0 -show_entries stream=width,height,r_frame_rate -show_entries format=duration,nb_frames -of default=noprint_wrappers=1:nokey=1 " + path;
	cout << cmd.c_str() << endl;
	//string cmd="ffprobe -v error -of flat=s=_ -select_streams v:0 -show_entries stream=width,height,nb_frames -of default=noprint_wrappers=1:nokey=1 "+path;
	string cmd2 = "ffprobe - select_streams v - show_streams" + path + " 2> NUL";

#ifdef __linux__
	FILE *stream = popen(cmd.c_str(), "r");
#else 
	FILE *stream = _popen(cmd.c_str(), "r");
#endif
	char buffer[50];
	fgets(buffer, 10, stream);
	data->width = atoi(buffer);
	fgets(buffer, 10, stream);
	data->height = atoi(buffer);
	fgets(buffer, 20, stream);
	string tmp = buffer;
	int pos = tmp.find('/');
	int fps1 = atoi(buffer);
	double fps2 = atoi(tmp.substr(pos + 1).c_str());
	double fps = fps1 / fps2;
	cout << fps << endl;
	fgets(buffer, 20, stream);
	//cout << buffer << endl;


	double len = atof(buffer);

	cout << len*fps << endl;
	data->frame_count = len*fps;
	data->size = data->width*data->height;
	//else data->frame_count = 3121;//181250; // 7100;//3121;//1359;//7192;
	return data;
}
FILE * startFFmpeg(string path) {
	FILE *stream;
#ifdef __linux__
	string cmd = "ffmpeg -i " + path + " -f image2pipe -pix_fmt yuv420p -vcodec rawvideo - 2>/dev/null";
	cout << cmd << endl;
	stream = popen(cmd.c_str(), "r");
#else 
	string cmd = "ffmpeg -i " + path + " -f image2pipe -threads 3  -pix_fmt yuv420p -vcodec rawvideo - 2>NUL";
	//-c:v h264_qsv
	stream = _popen(cmd.c_str(), "rb");
#endif
	cout << cmd.c_str() << endl;


	return stream;
}



double ** countMetric(FILE ** streams, FILE * ref, int files_count, PictureData * frame, string type, double ** results) {

	//double ** results = new double*[files_count];
	unsigned char *** data = new unsigned char **[files_count];
	unsigned char ** dataRef = new unsigned char *[CHUNK_SIZE];
	unsigned char * dataTrash = new unsigned char[frame->width*frame->height / 2];
	int frames = frame->frame_count;
	int rec;
	for (int k = 0; k < files_count; k++) {
		data[k] = new unsigned char *[CHUNK_SIZE];
		//results[k] = new double [frame->frame_count];
		for (int j = 0; j < CHUNK_SIZE; j++) {
			data[k][j] = new unsigned char[frame->width*frame->height];

		}
	}
	for (int j = 0; j < CHUNK_SIZE; j++) {
		dataRef[j] = new unsigned char[frame->width*frame->height];
	}

	for (int i = 0; i < frame->frame_count / CHUNK_SIZE; i++) {
		for (int j = 0; j < CHUNK_SIZE; j++) {
			rec = fread(dataRef[j], 1, frame->width*frame->height, ref);
			if (rec != frame->width*frame->height) {
				cout << "error" << endl;
				return NULL;
			}
			rec = fread(dataTrash, 1, frame->width*frame->height / 2, ref); //when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
			if (rec != frame->width*frame->height / 2) {
				cout << "error" << endl;
				return NULL;
			}
			for (int k = 0; k < files_count; k++) {
				int rec = fread(data[k][j], 1, frame->width*frame->height, streams[k]);
				if (rec != frame->width*frame->height) {
					cout << "error" << endl;
					return NULL;
				}
				rec = fread(dataTrash, 1, frame->width*frame->height / 2, streams[k]); //when using yuv, first 2/3 of the picture are Lumma, others are UV which we do not evaluate
				if (rec != frame->width*frame->height / 2) {
					cout << "error" << endl;
					return NULL;
				}
			}


		}
		omp_set_num_threads(CHUNK_SIZE);
		for (int k = 0; k < files_count; k++) {
			#pragma omp parallel for 
			for (int j = 0; j < CHUNK_SIZE; j++) {

				if (string(type) == string("SSIM")) results[k][j + i*CHUNK_SIZE] = countSSIM(dataRef[j], data[k][j], frame->width*frame->height, frame->width);

				else results[k][j + i*CHUNK_SIZE] = countPSNR(dataRef[j], data[k][j], frame->width*frame->height);
				//cout << j+i * CHUNK_SIZE << " " << results[k][j+i*CHUNK_SIZE] << endl;
			}
		}
	}
	for (int j = 0; j < frame->frame_count % CHUNK_SIZE; j++) {
		rec = fread(dataRef[j], 1, frame->width*frame->height, ref);
		fread(dataTrash, 1, frame->width*frame->height/2, ref);
		
		//fseek(ref, frame->width*frame->height / 2, SEEK_CUR); //skip others except Y channel
		if (rec != frame->width*frame->height/2) {
			cout << "error" << endl;
			return NULL;
		}
	}
	for (int k = 0; k < files_count; k++) {
		for (int j = 0; j < frame->frame_count % CHUNK_SIZE; j++) {
			rec = fread(data[k][j], 1, frame->width*frame->height, streams[k]);
			fread(dataTrash, 1, frame->width*frame->height/2, streams[k]);
			
			//fseek(streams[k], frame->width*frame->height / 2, SEEK_CUR); //skip others except Y channel
			if (rec != frame->width*frame->height) {
				cout << "error" << endl;
				return NULL;
			}

			if (string(type) == string("SSIM"))  results[k][frame->frame_count - frame->frame_count % CHUNK_SIZE + j] = countSSIM(dataRef[j], data[k][j], frame->width*frame->height, frame->width);
			else results[k][frame->frame_count - frame->frame_count % CHUNK_SIZE + j] = countPSNR(dataRef[j], data[k][j], frame->width*frame->height);
			//cout << frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j << " " << results[frame2->frame_count - frame2->frame_count % CHUNK_SIZE + j] << endl;
		}
	}

	//delete frame->data;
	//delete frame;
	return results;
}

int readFromFile(unsigned char *& data, int count, FILE * file) {
	int	rec = fread(data, 1, count, file);
	if (rec != count) {
		cout << "Error while reading from file" << endl;
		exit(-1);
	}
	return rec;
}


int main(int argc, char ** argv) {

	string reference;
	string file1, file2;
	string type;
	int gpu = 0;
	string  * files = new string[MAX_FILES];
	int files_count = 0;
	if (argc < 6) { // Check the value of argc. If not enough parameters have been passed, inform user and exit.
		cout << argc << endl;
		cout << "Usage is -r <reference file> -in <first video to compare> -in <second video to compare> [-type]\n"; // Inform the user of how to use the program
																													 //std::cin.get();
		exit(0);
	}
	else { // if we got enough parameters...

		//std::cout << argv[0];
		for (int i = 1; i < argc; i++) { /* We will iterate over argv[] to get the parameters stored inside.
										 * Note that we're starting on 1 because we don't need to know the
										 * path of the program, which is stored in argv[0] */
			if (i + 1 != argc) { // Check that we haven't finished parsing already
				if (string(argv[i]) == string("-r")) {
					// We know the next argument *should* be the filename:
					reference = argv[i + 1];
					//std::cout << reference << endl;
				}
				else if (string(argv[i]) == string("-in")) {
					files[files_count] = string(argv[i + 1]);
					//cout << files[files_count].c_str() << endl;
					files_count++;
				}
				/*else if (string(argv[i]) == string("-in2")) {
				file2 = string(argv[i + 1]);
				}*/
				else if (string(argv[i]) == string("-type")) {
					type = string(argv[i + 1]);

				}
				
				else {
					//cout << "Not enough or invalid arguments, please try again.\n";
					//Sleep(2000);
					//exit(0);
				}
				//std::cout << argv[i] << " ";
			}
			else if (i != argc){
				if (string(argv[i]) == string("CUDA")) { //we will use CUDA computation
					gpu = 1;
				}
			}
		}
	}

	const int MAX_BUFFER = 2048000;
	PictureData * frame;
	FILE ** streams;
	FILE * ref;
	streams = new FILE *[files_count];
	double ** results = new double *[files_count];
	frame = getVideoInfo(reference);
	ref = startFFmpeg(reference);
	for (int i = 0; i < files_count; i++) {
		frame = getVideoInfo(files[i]);
		streams[i] = startFFmpeg(files[i]);
		results[i] = new double[frame->frame_count];
	}


	int rec;
	double *sum = new double[files_count];
	int * frames = new int[files_count];
	for (int i = 0; i < files_count; i++) {
		frames[i] = frame->frame_count;
		sum[i] = 0;
	}
	if (gpu == 1) {
		if (string(type) == string("STVSSIM")) {
			cout << "stvssim CUDA" << endl;
			countMetricSTVSSIM_CUDA(streams, ref, files_count, frame, results, frames);
		}
		else {
			cout << "SSIM CUDA" << endl;
			countCUDA(streams, ref, files_count, frame, type, results);
		}
	}

	else if (string(type) == string("STVSSIM")) {
		cout << "stvssim CPU" << endl;
		countMetricSTVSSIM(streams, ref, files_count, frame, results, frames);
		delete streams; //?
		
	}
	else {
		cout << "SSIM/PSNR CPU" << endl;
		countMetric(streams, ref, files_count, frame, type, results); //SSIM, PSNR
		
	}

	for (int j = 0; j < files_count; j++) {
		cout << "input file number: " << j << endl;
		for (int i = 0; i < frames[j]; i++) {
			cout << i << " " << results[j][i] << endl;
			if (isfinite(results[j][i]))
				sum[j] += results[j][i];
			else frames[j]--;
		}

	}
	for (int i = 0; i < files_count; i++) {
		cout << "AVG: " << sum[i] / frames[i] << endl;
		qsort(results[i], frames[i], sizeof(double), compare);
		cout << "Median: " << results[i][frames[i] / 2] << endl;
	}
	//_CrtDumpMemoryLeaks();

}
